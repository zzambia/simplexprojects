#include "hip/hip_runtime.h"
#include "simplex.cuh"
#include<omp.h>
#include "iostream"

//LPC with Stream
//Method : To determine Pivot Column i.e., the entering variable we use the most negative value approach
//Race Condition completely avoided
//Both Reduction implemented- One for finding Pivot column and the other for finding Pivot Row
//Implemented Reduction. But without Streams.


//__global__ void mykernel(float *S_MAT, int S_row, int S_col, float *Result, int S_N, int *S_Sel, float *R_data, int *R_index) {
//1st Method : Most Negative Value approach
__global__ void mykernelCurrent(float *S_MAT, int S_row, int S_col, float *Result,int S_N, float *R_data, int *R_index,int offset_res) {
	int index = offset_res + blockIdx.x;
	if (index < (offset_res + S_N)) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		__shared__ float col1[1024];	//pivotColumn
		/*************/
		if (threadIdx.x == 0) {
			c = false;
			rm = 0;
			row = -1;		//pivotRow
			pivotCol = -1;
		}
		__syncthreads();
		while (!c) {
			__syncthreads();
			int Last_row = S_row - 1;
			__syncthreads();
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			__shared__ int notEntered;
//not used			__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
//not used				minValue = 0;
				newpivotcol = -1;
				notEntered = 1;
				c = true;
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			// Since keeping limit only upto (S_col - 1) which is not equal to BLOCK_SIZE creates problem
			// in using syncthreads() inside Reduction for-loop so use all threads(all R_data)
			//int data_size = (S_col - 1) - 2;
			int data_size = blockDim.x;
			tid = threadIdx.x;
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) {
				//tid = threadIdx.x - 2;//here my tid should be from 0 to (evenThreadId - 1)
				//int j = threadIdx.x;//is the actual column/index number less than (S_col - 1)
				//printf("Data_size = %d ", data_size);
				temp_index = Last_row + tid * S_row + base;	//avoiding re-computation
				R_data[tid + R_base] = S_MAT[temp_index];	//	g_data[i];
				R_index[tid + R_base] = tid;//tid; should be the real index of the data
			} else {
				R_data[tid + R_base] = INT_MAX;	//	g_data[i];
				R_index[tid + R_base] = tid;	//tid;
			}
			__syncthreads();//here will have all values in shared memory from 0 to BLOCK_SIZE

			tid = threadIdx.x;
			for (i = (data_size / 2); i > 0;) {
				if (tid < i) {
					//	if ((R_data[tid] >= R_data[tid + s]) && ((R_data[tid + s] < 0) && (R_data[tid] < 0))){
					//(R_data[tid + R_base] < 0) && (R_data[tid + R_base + i] < 0)&&
					if (R_data[tid + R_base] > R_data[tid + R_base + i]) { //is right-side value small?
						if (R_data[tid + R_base + i] <= -0.000001) {	//only if the value on the right-side is -ive
							R_data[tid + R_base] = R_data[tid + R_base + i];//put the smaller value to left-side
							R_index[tid + R_base] = R_index[tid + R_base + i];

							//notEntered = false;  //race condition avoided
						//	notEntered = 0;  //race condition avoided
							int local_notEntered;
							local_notEntered = *(volatile int*) &notEntered;
							atomicCAS(&notEntered, local_notEntered, 0);

						}
					}
				}
				/*if (tid == 0)
				 printf("Data_size = %d ", i);*/
				__syncthreads();
				i >>= 1;
				if ((i != 1) && (i % 2) != 0) {	//if s is odd
					i = i + 1;
				}
			}
			// if (notEntered == false && tid == 2) { // tid==0 is always true if minValue is still -1 then what?
			if (threadIdx.x == 0) { // tid==0 is always true if minValue is still -1 then what?
				if (notEntered == false) {
//not used					minValue = R_data[R_base];
					newpivotcol = R_index[R_base];
					//printf("\nminValue = %f newpivotcol = %d ", minValue,newpivotcol);
				}
			}
			__syncthreads(); //waiting for all threads to have same newpivotcol value
			if (newpivotcol == -1) {//All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2;
			} else {
				// ********** Second Reduction Process ******
				//in order to avoid global memory transfer:: Using the same R_data and R_index global memory
				__shared__ float row_min;
				__shared__ int row_num;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
				//if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
				if (threadIdx.x < Last_row) {
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					if ((S_MAT[temp_index2] > 0) && (S_MAT[temp_index1] > 0)) {
						R_data[k1 + R_base] = S_MAT[temp_index1]
								/ S_MAT[temp_index2];
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //here have all values from 0 to BLOCK_SIZE
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size
				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] >= R_data[indexValue2 + s]) {
							R_data[indexValue2] = R_data[indexValue2 + s];
							R_index[indexValue2] = R_index[indexValue2 + s];
							//notEntered2 = false;
							//notEntered2 = 0;//check using atomic
							int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);

						}
					}
//					if (tid == 0)
//						printf("Data_size = %d ", s);
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						//printf("R_Data = %f R_Index = %d", R_data[R_base], R_index[R_base]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						//if (notEntered2 == true) {
						//return -1;
						//printf("%f ", R_data[R_base]);
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						//}else {
						//return row_num;
						//printf("%f %d ", row_min, row_num);
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************

			//col = S_Sel[index];
			//col = *S_Sel;
			col = pivotCol;
			if (row > -1) {
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					if (tid == remember[tid - 2]) {
						temp_index = (S_row - 1) + (tid * S_row) + base; //avoiding re-computation
						S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}
				}		//Data Parallel section 1 done
				__syncthreads();
				tid = threadIdx.x;
				//if (threadIdx.x >= 0 && threadIdx.x < S_row) {
				if (threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();

				unsigned int temp_row_base = row + base;//avoiding re-computation
				S_MAT[temp_row_base + S_row] =
						S_MAT[temp_row_base + col * S_row];
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[row + base] = col - 1;//now temp_row_base is not required
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = S_MAT[temp_index] / S_MAT[row_base + S_row];
				}		//Data Parallel section 3 done
				__syncthreads();
				//printf("Row here = %d",row);
				tid = threadIdx.x;
				//if (threadIdx.x >= 0 && threadIdx.x < S_row) {
				if (threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					/*for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							S_MAT[temp_index] = S_MAT[temp_index]
									- (col1[tid] * S_MAT[row + temp_index1]);
						} else {
							break;
						}
					}*/


					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							float zeroTemp;
							zeroTemp = col1[tid] * S_MAT[row + temp_index1];
							S_MAT[temp_index] = S_MAT[temp_index] - zeroTemp;
						} else {
							break;
						}
					}

				}	//Data Parallel section 4 done
				__syncthreads();

				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				//tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[((S_row - 1) + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

			} else if (row == -1) {
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col;
					rm++;
				}
				__syncthreads();

				temp_index = (S_row - 1) + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive
				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				// tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[((S_row - 1) + i * S_row) + base] < 0)) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}
		} //end of while
		__syncthreads();
		if (threadIdx.x == 0) {
			//printf("Value = %f ",S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base]);
			Result[index] = S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base];
			//printf("blockIdx.x = %d   Result[index] = %f ",index,Result[index]);
		}
	}
}



//1st Method : Most Negative Value approach (//Works even for Large arguments)
__global__ void mykernelWorks(float *S_MAT, int S_row, int S_col, float *Result,
		int S_N, float *R_data, int *R_index) {
	//int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int index = blockIdx.x;
	if (index < S_N) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		__shared__ float col1[1024];	//pivotColumn
		/*************/
		if (threadIdx.x == 0) {
			c = false;
			rm = 0;
			row = -1;		//pivotRow
			pivotCol = -1;
		}
		__syncthreads();
		while (!c) {
			__syncthreads();
			int Last_row = S_row - 1;
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			__shared__ int notEntered;
			__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
				minValue = 0;
				newpivotcol = -1;
				notEntered = 1;
				c = true;
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			// Since keeping limit only upto (S_col - 1) which is not equal to BLOCK_SIZE creates problem
			// in using syncthreads() inside Reduction for-loop so use all threads(all R_data)
			//int data_size = (S_col - 1) - 2;
			int data_size = blockDim.x;
			tid = threadIdx.x;
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) {
				//tid = threadIdx.x - 2;//here my tid should be from 0 to (evenThreadId - 1)
				//int j = threadIdx.x;//is the actual column/index number less than (S_col - 1)
				//printf("Data_size = %d ", data_size);
				temp_index = Last_row + tid * S_row + base;	//avoiding re-computation
				R_data[tid + R_base] = S_MAT[temp_index];	//	g_data[i];
				R_index[tid + R_base] = tid;//tid; should be the real index of the data
			} else {
				R_data[tid + R_base] = INT_MAX;	//	g_data[i];
				R_index[tid + R_base] = tid;	//tid;
			}
			__syncthreads();//here will have all values in shared memory from 0 to BLOCK_SIZE

			tid = threadIdx.x;
			for (i = (data_size / 2); i > 0;) {
				if (tid < i) {
					//	if ((R_data[tid] >= R_data[tid + s]) && ((R_data[tid + s] < 0) && (R_data[tid] < 0))){
					//(R_data[tid + R_base] < 0) && (R_data[tid + R_base + i] < 0)&&
					if (R_data[tid + R_base] > R_data[tid + R_base + i]) { //is right-side value small?

						//if (R_data[tid + R_base + i] == -0.000000)
							 //R_data[tid + R_base + i] = 0.0;
						//This modification was required for large batch-size(1500) for LP dim 300 and above
						if (R_data[tid + R_base + i] <= -0.000001 ) {	//only if the value on the right-side is -ive
							R_data[tid + R_base] = R_data[tid + R_base + i];//put the smaller value to left-side
							R_index[tid + R_base] = R_index[tid + R_base + i];

							//notEntered = false;  //race condition avoided

							//notEntered = 0;  //race condition avoided
							int local_notEntered;
							local_notEntered = *(volatile int*) &notEntered;
							atomicCAS(&notEntered, local_notEntered, 0);

						}

					}
				}
				/*if (tid == 0)
				 printf("Data_size = %d ", i);*/
				__syncthreads();
				i >>= 1;
				if ((i != 1) && (i % 2) != 0) {	//if s is odd
					i = i + 1;
				}
			}
			// if (notEntered == false && tid == 2) { // tid==0 is always true if minValue is still -1 then what?
			if (threadIdx.x == 0) { // tid==0 is always true if minValue is still -1 then what?
				if (notEntered == false) {
					minValue = R_data[R_base];
					newpivotcol = R_index[R_base];
					//printf("\nminValue = %f newpivotcol = %d ", minValue,newpivotcol);
				}
			}
			__syncthreads(); //waiting for all threads to have same newpivotcol value
			if (newpivotcol == -1) {//All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2;
			} else {
				// ********** Second Reduction Process ******
				//in order to avoid global memory transfer:: Using the same R_data and R_index global memory
				__shared__ float row_min;
				__shared__ int row_num;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
				if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					//This modification was required for large batch-size(5000) for LP dim 300 and above
					if ((S_MAT[temp_index2] >= -0.000001) && (S_MAT[temp_index1] >= -0.000001)) {
						R_data[k1 + R_base] = S_MAT[temp_index1]/ S_MAT[temp_index2];
						//R_data[k1 + R_base] = roundf(((S_MAT[temp_index1]/ S_MAT[temp_index2])/1000000)*1000000);
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //here have all values from 0 to BLOCK_SIZE
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size
				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] >= R_data[indexValue2 + s]) {
							R_data[indexValue2] = R_data[indexValue2 + s];
							R_index[indexValue2] = R_index[indexValue2 + s];
							//notEntered2 = false;

							//notEntered2 = 0;
							int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);
						}
					}
//					if (tid == 0)
//						printf("Data_size = %d ", s);
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						//printf("R_Data = %f R_Index = %d", R_data[R_base], R_index[R_base]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						//if (notEntered2 == true) {
						//return -1;
						//printf("%f ", R_data[R_base]);
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						//}else {
						//return row_num;
						//printf("%f %d ", row_min, row_num);
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************

			//col = S_Sel[index];
			//col = *S_Sel;
			col = pivotCol;
			//printf("Row= %d col = %d\n",row,col);
			if (row > -1) {
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					if (tid == remember[tid - 2]) {
						temp_index = (S_row - 1) + (tid * S_row) + base; //avoiding re-computation
						//if (S_MAT[temp_index] == -0.0)
							//S_MAT[temp_index] = -1 * 0.0; //replacing back to original
						//else
							S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}
				}		//Data Parallel section 1 done
				__syncthreads();
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();

				unsigned int temp_row_base = row + base;//avoiding re-computation
				S_MAT[temp_row_base + S_row] =
						S_MAT[temp_row_base + col * S_row];
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[row + base] = col - 1;//now temp_row_base is not required
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = roundf(((S_MAT[temp_index] / col1[row])/1000000)*1000000);//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = S_MAT[temp_index] / S_MAT[row_base + S_row];
				}		//Data Parallel section 3 done
				__syncthreads();
				//printf("Row here = %d",row);
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							float zeroTemp;
							zeroTemp = col1[tid] * S_MAT[row + temp_index1];
							S_MAT[temp_index] = S_MAT[temp_index] - zeroTemp;
							//S_MAT[temp_index] = roundf(((S_MAT[temp_index] - zeroTemp)/1000000)*1000000);
							//if (S_MAT[temp_index] == -0.0)
								//S_MAT[temp_index]= 0.0;

						} else {
							break;
						}
					}
				}	//Data Parallel section 4 done
				__syncthreads();

				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				//tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[((S_row - 1) + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

			} else if (row == -1) {
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col;
					rm++;
				}
				__syncthreads();

				temp_index = (S_row - 1) + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				//if (S_MAT[temp_index] == -0.0)
					//S_MAT[temp_index] = 0;	//remembering by making positive
				//else
					S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive
				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				// tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[((S_row - 1) + i * S_row) + base] < 0)) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}

		} //end of while
		__syncthreads();

		if (threadIdx.x == 0) {
			//printf("Value = %f ",S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base]);
			Result[index] = S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base];
			//printf("\nResult Inside Kernel: %f \n",Result[index]);
		}
	}
}

//1st Method : Most Negative Value approach
__global__ void mykernel(float *S_MAT, int S_row, int S_col, float *Result,int S_N, float *R_data, int *R_index,int offset_res) {
	int index = offset_res + blockIdx.x;
	if (index < (offset_res + S_N)) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		__shared__ float col1[1024];	//pivotColumn
		/*************/
		if (threadIdx.x == 0) {
			c = false;
			rm = 0;
			row = -1;		//pivotRow
			pivotCol = -1;
		}
		__syncthreads();
		while (!c) {
			__syncthreads();
			int Last_row = S_row - 1;
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			__shared__ int notEntered;
			__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
				minValue = 0;
				newpivotcol = -1;
				notEntered = 1;
				c = true;
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			// Since keeping limit only upto (S_col - 1) which is not equal to BLOCK_SIZE creates problem
			// in using syncthreads() inside Reduction for-loop so use all threads(all R_data)
			//int data_size = (S_col - 1) - 2;
			int data_size = blockDim.x;
			tid = threadIdx.x;
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) {
				//tid = threadIdx.x - 2;//here my tid should be from 0 to (evenThreadId - 1)
				//int j = threadIdx.x;//is the actual column/index number less than (S_col - 1)
				//printf("Data_size = %d ", data_size);
				temp_index = Last_row + tid * S_row + base;	//avoiding re-computation
				R_data[tid + R_base] = S_MAT[temp_index];	//	g_data[i];
				R_index[tid + R_base] = tid;//tid; should be the real index of the data
			} else {
				R_data[tid + R_base] = INT_MAX;	//	g_data[i];
				R_index[tid + R_base] = tid;	//tid;
			}
			__syncthreads();//here will have all values in shared memory from 0 to BLOCK_SIZE

			tid = threadIdx.x;
			for (i = (data_size / 2); i > 0;) {
				if (tid < i) {
					//	if ((R_data[tid] >= R_data[tid + s]) && ((R_data[tid + s] < 0) && (R_data[tid] < 0))){
					//(R_data[tid + R_base] < 0) && (R_data[tid + R_base + i] < 0)&&
					if (R_data[tid + R_base] > R_data[tid + R_base + i]) { //is right-side value small?

						//if (R_data[tid + R_base + i] == -0.000000)
							 //R_data[tid + R_base + i] = 0.0;
						if (R_data[tid + R_base + i] <= -0.000001 ) {	//only if the value on the right-side is -ive
							R_data[tid + R_base] = R_data[tid + R_base + i];//put the smaller value to left-side
							R_index[tid + R_base] = R_index[tid + R_base + i];

							//notEntered = false;  //race condition avoided

							//notEntered = 0;  //race condition avoided
							int local_notEntered;
							local_notEntered = *(volatile int*) &notEntered;
							atomicCAS(&notEntered, local_notEntered, 0);

						}

					}
				}
				__syncthreads();
				i >>= 1;
				if ((i != 1) && (i % 2) != 0) {	//if s is odd
					i = i + 1;
				}
			}
			// if (notEntered == false && tid == 2) { // tid==0 is always true if minValue is still -1 then what?
			if (threadIdx.x == 0) { // tid==0 is always true if minValue is still -1 then what?
				if (notEntered == false) {
					minValue = R_data[R_base];
					newpivotcol = R_index[R_base];
					//printf("\nminValue = %f newpivotcol = %d ", minValue,newpivotcol);
				}
			}
			__syncthreads(); //waiting for all threads to have same newpivotcol value
			//		__syncthreads();	//here we have min and newpivotcol
			// ********* First Reduction Ends *************
			//  ******** Second Reduction Begins **********
			if (newpivotcol == -1) {//All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2;
			} else {
				// ********** Second Reduction Process ******
				//in order to avoid global memory transfer:: Using the same R_data and R_index global memory
				__shared__ float row_min;
				__shared__ int row_num;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
				if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					if ((S_MAT[temp_index2] > 0) && (S_MAT[temp_index1] > 0)) {
						R_data[k1 + R_base] = S_MAT[temp_index1]/ S_MAT[temp_index2];
						//R_data[k1 + R_base] = roundf(((S_MAT[temp_index1]/ S_MAT[temp_index2])/1000000)*1000000);
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //here have all values from 0 to BLOCK_SIZE
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size
				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] >= R_data[indexValue2 + s]) {
							R_data[indexValue2] = R_data[indexValue2 + s];
							R_index[indexValue2] = R_index[indexValue2 + s];
							//notEntered2 = false;

							//notEntered2 = 0;
							int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);
						}
					}
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						//printf("R_Data = %f R_Index = %d", R_data[R_base], R_index[R_base]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************
			col = pivotCol;
			if (row > -1) {
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					if (tid == remember[tid - 2]) {
						temp_index = (S_row - 1) + (tid * S_row) + base; //avoiding re-computation
						S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}
				}		//Data Parallel section 1 done
				__syncthreads();
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();

				unsigned int temp_row_base = row + base;//avoiding re-computation
				S_MAT[temp_row_base + S_row] =
						S_MAT[temp_row_base + col * S_row];
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[row + base] = col - 1;//now temp_row_base is not required
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//S_MAT[row_base + S_row];
				}		//Data Parallel section 3 done
				__syncthreads();
				//printf("Row here = %d",row);
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							float zeroTemp;
							zeroTemp = col1[tid] * S_MAT[row + temp_index1];
							S_MAT[temp_index] = S_MAT[temp_index] - zeroTemp;
						} else {
							break;
						}
					}
				}	//Data Parallel section 4 done
				__syncthreads();
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[((S_row - 1) + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

			} else if (row == -1) {
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col;
					rm++;
				}
				__syncthreads();

				temp_index = (S_row - 1) + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive

				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[((S_row - 1) + i * S_row) + base] < 0)) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}

		} //end of while
		__syncthreads();

		if (threadIdx.x == 0) {
			Result[index] = S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base];
		//	printf("\nResult Inside Kernel: %f \n",Result[index]);
		}
	}
}



// 2nd Method:: Random negative value
__global__ void mykernel2(float *S_MAT, int S_row, int S_col, float *Result,
		int S_N, float *R_data, int *R_index) {
	//int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int index = blockIdx.x;
	if (index < S_N) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		__shared__ float col1[1024];	//pivotColumn
		/*************/
		//if (threadIdx.x == 0) {
			c = false;
			rm = 0;
			row = -1;		//pivotRow
			pivotCol = -1;
		//}
		__syncthreads();
		while (!c) {
			//__syncthreads();
			int Last_row = S_row - 1;
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
				minValue = 0;
				newpivotcol = -1;
				c = true;
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			//for (int j = 2; j < S_col - 1; j++) {//only last row but all column
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) {
				int j = threadIdx.x;
				unsigned int temp_index1 = Last_row + j * S_row + base; //avoiding re-computation
				if (S_MAT[temp_index1] < minValue) {
					//minValue = S_MAT[temp_index1];
					newpivotcol = j; //"Any(Random) negative coefficient rule"
					/*
					int local_NewPivotCol;
					local_NewPivotCol = *(volatile int*) &newpivotcol;
					atomicCAS(&newpivotcol, local_NewPivotCol, j);
					*/
					/*
					 http://stackoverflow.com/questions/27616417/cuda-is-there-any-way-to-prevent-other-threads-from-changing-a-shared-or-global
					 if (atomicCAS(&newpivotcol, local_NewPivotCol, j)==local_NewPivotCol){
					 //this thread won the write
					 printf("Thread ID = %d ",threadIdx.x);
					 }*/
					//break;
				}
			}
			__syncthreads(); //here we have min and newpivotcol

			//  ******** Second Reduction Begins **********

			if (newpivotcol == -1) { //All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2;
			} else {
				// ********** Second Reduction Process ******
				//in order to avoid global memory transfer:: Using the same R_data and R_index global memory
				__shared__ float row_min;
				__shared__ int row_num;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
				if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					if ((S_MAT[temp_index2] > 0) && (S_MAT[temp_index1] > 0)) {
						R_data[k1 + R_base] = S_MAT[temp_index1]
								/ S_MAT[temp_index2];
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //here have all values from 0 to BLOCK_SIZE
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size
				/*if (threadIdx.x == 0) {
				 printf("\nR_data \n");
				 for (int x = 0; x < Last_row; x++)
				 printf("%f  ", R_data[x]);
				 printf("\nR_Index \n");
				 for (int x = 0; x < Last_row; x++)
				 printf("%d  ", R_index[x]);
				 printf("Data_size2 = %d ", data_size2);
				 }
				 __syncthreads();*/
				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] >= R_data[indexValue2 + s]) {
							R_data[indexValue2] = R_data[indexValue2 + s];
							R_index[indexValue2] = R_index[indexValue2 + s];
							//notEntered2 = false;
							notEntered2 = 0;
							/*int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);
							*/
						}
					}
//					if (tid == 0)
//						printf("Data_size = %d ", s);
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						//printf("R_Data = %f R_Index = %d", R_data[R_base], R_index[R_base]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						//if (notEntered2 == true) {
						//return -1;
						//printf("%f ", R_data[R_base]);
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						//}else {
						//return row_num;
						//printf("%f %d ", row_min, row_num);
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************

			//col = S_Sel[index];
			//col = *S_Sel;
			col = pivotCol;
			if (row > -1) {
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					if (tid == remember[tid - 2]) {
						temp_index = (S_row - 1) + (tid * S_row) + base; //avoiding re-computation
						S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}
				}		//Data Parallel section 1 done
				__syncthreads();
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();

				unsigned int temp_row_base = row + base;//avoiding re-computation
				S_MAT[temp_row_base + S_row] =
						S_MAT[temp_row_base + col * S_row];
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[row + base] = col - 1;//now temp_row_base is not required
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = S_MAT[temp_index] / S_MAT[row_base + S_row];
				}		//Data Parallel section 3 done
				__syncthreads();
				//printf("Row here = %d",row);
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							S_MAT[temp_index] = S_MAT[temp_index]
									- (col1[tid] * S_MAT[row + temp_index1]);
						} else {
							break;
						}
					}
				}	//Data Parallel section 4 done
				__syncthreads();

				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				//tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[((S_row - 1) + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

			} else if (row == -1) {
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col;
					rm++;
				}
				__syncthreads();

				temp_index = (S_row - 1) + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive
				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				// tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[((S_row - 1) + i * S_row) + base] < 0)) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}
		} //end of while
		__syncthreads();
		if (threadIdx.x == 0) {
			//printf("Value = %f ",S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base]);
			Result[index] = S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base];
		}
	}
}


__host__ Simplex::Simplex(unsigned int N_S) {
	number_of_LPs = N_S;
	M = 0;
	N = 0;
	c = 0;
	No_c = 0;
	/*unsigned int memSize = N_S * sizeof(float);
	 R = (float*) malloc(memSize);*/
}
//get status of particular simplex
__host__ int Simplex::getStatus(int n) {
	int s;
	for (int i = 0; i < C.size1(); i++) {
		if (i == (n - 1)) {
			if (R[i] == -1) {
				s = 6;	// 6 = Simplex Is Unbounded
			} else if (R[i] > 0) {
				s = 2;	// 2= Simplex has feasible and Optimal solution
			}
		}
	}
	return s;

}	//get status of particular simplex

//get the No of simplex the object is ruuning on GPU
__host__ int Simplex::getNo_OF_Simplx() {
	return C.size1();
}	//get the No of simplex the object is ruuning on GPU

//get the result of all simplex
__host__ std::vector<float> Simplex::getResultAll() {

	std::vector<float> Res(C.size1());
	for (int i = 0; i < C.size1(); i++) {
		//std::cout<<"No error here!!!\n";
		Res[i] = R[i];
	}
	return Res;
}

//get the result of all simplex

__host__ float Simplex::getResult(int n) {
	// get result of particular simplex
	float r;
	for (int i = 0; i < C.size1(); i++) {
		if (i == (n - 1)) {
			r = R[i];
		}
	}
	return r;
}	// get result of particular simplex

__host__ std::vector<int> Simplex::getStatusAll() {

	std::vector<int> Status(C.size1());
	for (int i = 0; i < C.size1(); i++) {
		if (R[i] == -1)
			Status[i] = 6;
		else
			Status[i] = 2;
	}
	return Status;
}	//get the status of all simplex

__host__ void Simplex::setConstratint(math::matrix<double> A,
		std::vector<double> B) {
	int N_S = number_of_LPs;
	orig_CoefficientMatrix = A;
	BoundValue = B;
	int No_O = A.size2();
	int No_C = A.size1();
	M = No_C + 1;
	N = No_O + 3 + No_C;
	c = 1 + No_O;
		//MAT = (float *) calloc(N_S * M * N, sizeof(float));
		unsigned int memSize = N_S * M * N * sizeof(float);
		hipError_t err;
		err = hipHostMalloc(&MAT, memSize);//Pinned memory Syntax:: hipHostMalloc(&h_ptr,bytes);
		//printf("CUDA hipHostMalloc-- MAT: %s\n", hipGetErrorString(err));
		hipMemset(MAT, 0, memSize);	//initializing all elements to zero
#pragma omp parallel for
	for (int s = 0; s < N_S; s++) {
		for (int i = 0; i < M - 1; i++) {
			for (int j = 0; j < N; j++) {
				if (j == 0) {
					MAT[(int) ((i + j * M) + (M * N * s))] = c + i;
				}
				else if (j > 1) {
					if (j < (No_O + 2)) {
						MAT[(int) ((i + j * M) + (M * N * s))] = (float) A(i,
								j - 2);
					} else if (j == N - 1) {
						MAT[(int) ((i + j * M) + (M * N * s))] = (float) B[i];
					} else if (j < N - 1) {
						MAT[(int) ((i + (No_O + 2 + i) * M) + (M * N * s))] = 1;
					}
				}
			}
		}
	}
	//std::cout<<"Constraints Setting Over!!!\n";
}	

__host__ void Simplex::ComputeLP(math::matrix<float> &C1,unsigned int number_of_streams ) {

	hipError_t err;
	unsigned int threads_per_block;	//Maximum threads depends on CC 1.x =512 2.x and > = 1024
	unsigned int number_of_blocks;//depends on our requirements (better to be much more than the number of SMs)

	int device;
	hipDeviceProp_t props;
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device);

	int No_C = orig_CoefficientMatrix.size1();
	C = math::matrix<float>(C1);

	int N_S = C.size1();
	unsigned int memSize = N_S * sizeof(float);
	//R = (float*) malloc(memSize);
	err = hipHostMalloc((void**)&R, memSize);	//PINNED Memory	 //hipHostMalloc((void**)&a, bytes) );      // host pinned
	//printf("CUDA hipHostMalloc-- R: %s\n", hipGetErrorString(err));
	int No_O = C.size2();
	M = No_C + 1;
	N = No_O + 3 + No_C;
	int N_C = No_C;
	c = 1 + No_O;
	//float sum = 0;
#pragma omp parallel for
	for (int s = 0; s < N_S; s++) {
		for (int i = M - 1; i < M; i++) {
			for (int j = 2; j < N; j++) {
				if (j < 2 + No_O) {
					MAT[(int) ((i + j * M) + (M * N * s))] = -C(s, j - 2);
				}	
			}
		}
	}
	std::vector<int> rem;
	for (int i = 0; i < N_C; i++) {
		//std::cout<<BoundValue[i]<<"\n";
		if (BoundValue[i] < 0) {
			rem.push_back(i);
			
		}
	}

	//std::cout<<"C= "<< rem.size()<<"\n";
	int nc = N + rem.size();
	threads_per_block = 32 * (nc / 32) + 32; //if count equal 0 than nc=N so works for for Model
	if (threads_per_block > props.maxThreadsPerBlock) //Assuming maximum threads supported by CC is 1024
		threads_per_block = props.maxThreadsPerBlock;
	int offset;
	int *R_index;	//reduction data
	float *R_data;	//reduction index
	err = hipMalloc((void **) &R_data, C1.size1() * threads_per_block * sizeof(float));//C1.size1() * 96 being the maximum threads
	err = hipMalloc((void **) &R_index,C1.size1() * threads_per_block * sizeof(int));//C1.size1() being the number of LPs
	err = hipMalloc((void **) &G_R, N_S * sizeof(float));//Doing it here for the First Time
									// eg 	 hipMalloc((void**)&d_a, bytes) ); // device
//	printf("CUDA malloc R_index: %s\n", hipGetErrorString(err));
//	std::cout << "Number of threads per block = " << threads_per_block << "\n";
	if (rem.size() > 0) {
		;
	}
	else {
			err = hipMalloc((void **) &G_MAT, (N_S * M * N * sizeof(float)));
			//printf("CUDA malloc G_MAT : %s\n", hipGetErrorString(err));
			// **** Begin of Stream Processing *******
			//Using Asynchronous Memory copy:: needs //MAT to be a PINNED memory
			int num_streams = number_of_streams;//number of streams desired to create ::Note check for odd numbers
			int Each_LP_size = M * N;	// * sizeof(float);
			int num_LPs_perStream;
			bool equal_stream = true;
			if (N_S % num_streams == 0) {
				num_LPs_perStream = (N_S / num_streams);
				equal_stream = true;
			} else {
				num_LPs_perStream = (N_S / num_streams);//last stream will not be of the same size
				num_streams = num_streams + 1;//one extra streams.where nos of LPs to be solved will be less;
				equal_stream = false;
			}
			hipStream_t stream[num_streams];
			hipError_t result;

			//Creation of Streams
			for (int i = 0; i < num_streams; i++) {
				result = hipStreamCreate(&stream[i]);
			}

			//err = hipMemcpy(G_MAT, MAT, (N_S * M * N * sizeof(float)),hipMemcpyHostToDevice);
			//Stream -- memcopy Host to Device
		//	std::cout << "\nNumber of LPs_perStream = " << num_LPs_perStream << std::endl;
			unsigned int lastBlock_size;
			if (equal_stream == false) {
				lastBlock_size = N_S - (N_S / (num_streams - 1)) * (num_streams - 1);//LAST Stream Size
			//	std::cout << "\nAmit Last Block size (LPs is )= " << lastBlock_size<< std::endl;
			}

			for (int i = 0; i < num_streams; i++) {
				if (equal_stream == false && i == (num_streams - 1)) {//last stream
					int offset = i * Each_LP_size * lastBlock_size;	//for memory copy
					hipMemcpyAsync(&G_MAT[offset], &MAT[offset], (lastBlock_size * M * N * sizeof(float)), hipMemcpyHostToDevice, stream[i]);
				} else {
					int offset = i * Each_LP_size * num_LPs_perStream;//for memory copy
					hipMemcpyAsync(&G_MAT[offset], &MAT[offset], (num_LPs_perStream * M * N * sizeof(float)), hipMemcpyHostToDevice, stream[i]);
				}
			}
			//mykernel<<<N_S, threads_per_block>>>(G_MAT, M, N, G_R, N_S, G_Sel, R_data, R_index);
			//	mykernel<<<N_S, threads_per_block>>>(G_MAT, M, N, G_R, N_S, R_data, R_index);
			//std::cout << "Before Kernel Call!!!" << std::endl;
			//Stream -- Kernel
			for (int i = 0; i < num_streams; i++) {
				if (equal_stream == false && i == (num_streams - 1)) {//last stream
					int offset_res = i * lastBlock_size;//for result here offset_res is a pointer to the LP number
					//mykernel<<<num_LPs_perStream, 256, 0, stream[i]>>>(G_MAT, M, N, G_R, G_Sel, num_LPs_perStream, offset_res);
					mykernel<<<lastBlock_size, threads_per_block, 0, stream[i]>>>(G_MAT, M, N, G_R, lastBlock_size, R_data, R_index, offset_res);
				} else {
					int offset_res = i * num_LPs_perStream;	//for result here offset_res is a pointer to the LP number
					//mykernel<<<num_LPs_perStream, 256, 0, stream[i]>>>(G_MAT, M, N, G_R, G_Sel, num_LPs_perStream, offset_res);
				//	std::cout<<"Kernel Called!!!\n";
					mykernel<<<num_LPs_perStream, threads_per_block, 0, stream[i]>>>(G_MAT, M, N, G_R, num_LPs_perStream, R_data, R_index, offset_res);
				//	std::cout<<"Kernel Finished!!!\n";
				//	std::cout<<"this kernel\n";
				}
			}
		//	std::cout << "After Kernel Call!!!" << std::endl;
		//	hipDeviceSynchronize();//removed as hopping that hipFree will handle it
			//err = hipMemcpy(R, G_R, N_S * sizeof(float),hipMemcpyDeviceToHost);

			//Stream -- memcopy Device to Host
			for (int i = 0; i < num_streams; i++) {
				//if (equal_stream == false && i == (num_streams - 1)) {//last stream
					//int offset_res = i * lastBlock_size;//for result here offset_res is a pointer to the LP number
					//hipMemcpyAsync(&R[offset_res], &G_R[offset_res],(lastBlock_size * sizeof(float)),hipMemcpyDeviceToHost, stream[i]);
				//} else {
					int offset_res = i * num_LPs_perStream;	//for result here offset_res is a pointer to the LP number
				//	std::cout<<"offset_res = "<<offset_res<<std::endl;
				//	std::cout<<"Memcopy started!!!\n";
					hipMemcpyAsync(&R[offset_res], &G_R[offset_res],(num_LPs_perStream * sizeof(float)),
							hipMemcpyDeviceToHost, stream[i]);
				//	std::cout<<"Memcopy Finished!!!\n";
				//	printf("CUDA memcopyAsync G_R : %s\n", hipGetErrorString(err));
				//}
			}
			//std::cout<<"Memcopy End of Memory Copy!!!\n";
			// **** End of Stream Processing *******
			//printf("CUDA memcpy G_R: %s\n", hipGetErrorString(err));
			//	std::cout << "Testing: R[0] = " << R[0] << std::endl;

				/*for (int i = 0; i < num_streams; ++i)
				    hipStreamDestroy(stream[i]);*/
		}
	//std::cout<<"Before hipFree command 1 !!!\n";
		//hipFree(R_index);	//Only to synchronise with the cudamemcpy
		//hipFree(R_data);	//Only to synchronise with the cudamemcpy
		hipFree(G_MAT); //OK required
	//std::cout<<"Before hipFree command 2 !!!\n";
		//hipHostFree(MAT); //should not be used
		hipFree(G_R); //OK required
		//hipHostFree(R);	//This is needed to avoid Segmentation fault error
	//	std::cout << "N_S after = " << R[1] << std::endl;
	//	std::cout<<"After hipFree command!!!\n";
}

