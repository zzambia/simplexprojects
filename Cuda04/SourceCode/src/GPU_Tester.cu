#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <stdio.h>
#include <stdlib.h>
#include "simplex.cuh"
#include "math/glpk_lp_solver/glpk_lp_solver.h"
#include <vector>
#include "math/matrix.h"
#include <climits>
#include <iostream>
#include "boost/timer/timer.hpp"
#include "sys/time.h"

int main(int argc, char *argv[]) {
	unsigned int LP_size = 1, avg, dimension, stream;
	math::matrix<double> A;
	math::matrix<float> C, newC;
	std::vector<double> b;
	std::vector<float> result;
	std::vector<int> status_val;
	double Final_Time;


	if (argc > 1) {
		if (argc != 5) {		//1(ApplicationName) + 4 (Input Arguments)
			std::cout << "\nInsufficient Number of Arguments!!!\n";
			std::cout << "Correct Usages/Syntax:\n";
			std::cout<< "./ProjName --'Dimension'--'Average'--'Batch-size' --'Streams'!!\n";
			std::cout<< "argument 1) Dimension -- select the dimension of LP to be solved\n";
			std::cout<< "argument 2) Average -- select the number of runs for average readings\n";
			std::cout<< "argument 3) Batch-size -- select the number of LPs to be solved\n";
			std::cout<< "argument 4) Streams -- 0 for no streaming, 1 or 10 or 'n' to select the number of streams \n";
			return 0;
		} else {
			unsigned int num;
			dimension = atoi(argv[1]);
			num = atoi(argv[2]);
			avg = num;
			LP_size = atoi(argv[3]);
			stream = atoi(argv[4]);
		}
	}

	glpk_lp_solver lp;
	unsigned int status = 0;
	//**** Creating Random Lp's Any x Any ****
	while (status != 5) {
		A.resize(dimension, dimension);
		b.resize(dimension);
		for (unsigned int j = 0; j < dimension; j++) {
			for (unsigned int k = 0; k < dimension; k++) {
				A(j, k) = rand() % (k + 10) + 1;

			}
			b[j] = (rand() % (j + 1) + (10 + j));
		}

		//** Setting current Lp to GLPK
		lp.setMin_Or_Max(2);
		lp.setConstraints(A, b, 1);
		status = lp.TestConstraints();
	}
	/*
	 * Experimental Note:
	 *  We generate randomly the first LP problem (the matrix A and vector b) and then generate
	 *  randomly the objective function(s). For the ease generation process we take all the
	 *  LP problems to be the same LP generated above. However, for each of this LPs we generate
	 *  randomly different objective functions.
	 *
	 *  Also, to record the average time we ignore the first reading taken in GPU as it does not
	 *  reflect the correct computation time because for the first GPU call it also include an
	 *  extra overhead of GPU initialization time.
	 */

	C.resize(LP_size, dimension);
	for (unsigned int i = 0; i < C.size1(); i++) {
		for (unsigned int j = 0; j < C.size2(); j++) {
			C(i, j) = rand() % (j + 1) + 1;
		}
	}

	//Computation for CPU ie GLPK
	double sum = 0.0;
	double wall_clock, return_Time;
	boost::timer::cpu_timer tt1,tt2;	//tt1 -- Variable declaration

	std::cout << "\n*****GLPK RESULT*****\n";
	std::vector<double> dir(dimension);

	//***** MODEL SELECTION *****
	double res = 0.0;
	double batchTime = 0.0, AvgBatchTime = 0.0;
	std::vector<double> resul(C.size1());
	for (int i = 1; i <= avg; i++) {
		tt1.start();
		for (int i = 0; i < C.size1(); i++) {
			glpk_lp_solver mylp;
			mylp.setMin_Or_Max(2);
			for (int j = 0; j < dimension; j++) {
				dir[j] = C(i, j);
			}
			mylp.setConstraints(A, b, 1); //this function actually determines independent LP in GLPK
			res = mylp.Compute_LLP(dir); //We consider every dir an independent LP problem
			resul[i] = res;
		}
		tt1.stop();
		wall_clock = tt1.elapsed().wall / 1000000; //convert nanoseconds to milliseconds
		return_Time = wall_clock / (double) 1000; //convert milliseconds to seconds
		batchTime = return_Time; //convert nanoseconds to milliseconds
		AvgBatchTime = AvgBatchTime + batchTime;
	}

	//std::cout << "Last Result:" << res << "\t";
	Final_Time = AvgBatchTime / avg;
	std::cout << "\nNumber of Simplex Solved = " << C.size1() << std::endl;
	std::cout << "\nBoost Time taken:Wall  (in Seconds) GLPK:= " << (double) Final_Time << std::endl;

	std::cout << "\n*****GPU RESULT*****\n";
	sum = 0.0;
	for (unsigned int i = 0; i <= avg; i++) {
		Simplex s(C.size1());
		s.setConstratint(A, b);
		tt1.start();
		s.ComputeLP(C, stream);
		tt1.stop();
		wall_clock = tt1.elapsed().wall / 1000000; //convert nanoseconds to milliseconds
		return_Time = wall_clock / (double) 1000; //convert milliseconds to seconds

		std::cout << "Iter = " << i << " Time = " << return_Time<< std::endl;
		if (i != 0) {
			wall_clock = tt1.elapsed().wall / 1000000; //convert nanoseconds to milliseconds
			return_Time = wall_clock / (double) 1000; //convert milliseconds to seconds
			//std::cout << "Iter = " << i << " Time = " << return_Time<< std::endl;
			sum = sum + return_Time; //convert nanoseconds to milliseconds
		}
		result = s.getResultAll();
	}

	Final_Time = sum / avg;
	std::cout << "\nNumber of Simplex Solved = " << C.size1() << std::endl;
	std::cout << "\nBoost Time taken:Wall  (in Seconds) GPU:= " << (double) Final_Time << std::endl;
	std::cout << "\n**Answer_Of_All_Simplex**\n";

	int max = 5;	//Verifying results of only first 5 LPs.
	if (LP_size < max)
		max = LP_size;

	std::cout << "\nVERIFICATION FOR CORRECTNESS\n";
	for (int i=0;i<max;i++) {
		std::cout << "GLPK: " << resul[i] << " || GPU: " << result[i] << std::endl;
	}
	return 0;
}

