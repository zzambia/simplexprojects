#include "hip/hip_runtime.h"
#include "simplex.cuh"
#include<omp.h>
#include "iostream"
#include <math.h>


//1st Method : Most Negative Value approach
__global__ void mykernel(float *S_MAT, int S_row, int S_col, float *Result, int S_N, float *R_data, int *R_index) {
	//int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int index = blockIdx.x;
	if (index < S_N) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
//		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		//Debug ------ Initializing ------------
		remember[threadIdx.x]= 7777; //Since our maximum value might be 1024 for very large LP. But depending upon dim, that many remember[] will be assigned
		//-------------------
		//int Last_row = S_row - 1;//Amit now this should be 2nd last row
		int Last_row = S_row - 2;//Amit now this should be 2nd last row

		__shared__ float col1[1024];	//pivotColumn
		/*************/
		if (threadIdx.x == 0) {
//			printf("sizeof(float) =%d ",sizeof(float));
//			printf("\nsizeof(int) =%d \n", sizeof(int));
			c = false;
			rm = 0;
			row = -1;		//pivotRow
//			pivotCol = -1;
		}
		__syncthreads();
		while (!c) {
			__syncthreads();
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			__shared__ int notEntered;
			__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
				row = -1;		//pivotRow
				minValue = 0;
				newpivotcol = -1;
				notEntered = 1;
				c = true;
			/*	printf("Printing Remembered\n");
				for (int tt=0;tt<1024;tt++)
					printf("Remember[%d] = %d \n",tt, remember[tt]);
				printf("\n");*/
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			// Since keeping limit only upto (S_col - 1) which is not equal to BLOCK_SIZE creates problem
			// in using syncthreads() inside Reduction for-loop so use all threads(all R_data)
			//int data_size = (S_col - 1) - 2;
			int data_size = blockDim.x;
			tid = threadIdx.x;
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) { //find minimum from last row leaving last column
				//tid = threadIdx.x - 2;//here my tid should be from 0 to (evenThreadId - 1)
				//int j = threadIdx.x;//is the actual column/index number less than (S_col - 1)
				//printf("Data_size = %d ", data_size);
				temp_index = Last_row + tid * S_row + base;	//avoiding re-computation
				R_data[tid + R_base] = S_MAT[temp_index];	//	g_data[i];
				R_index[tid + R_base] = tid;//tid; should be the real index of the data
			} else {
				R_data[tid + R_base] = INT_MAX;	//	g_data[i];
				R_index[tid + R_base] = tid;	//tid;
			}
			__syncthreads();//here will have all values in shared memory from 0 to BLOCK_SIZE
			//Debug----
			/*if (threadIdx.x==0){
				printf("\n Data and Index");
				for (int x=0;x<blockDim.x;x++){
					printf("(%f, %d), ", R_data[R_base + x], R_index[R_base + x]);
				}
				printf("\n");
			}
			__syncthreads();*/
			//----- Verified correct copy of Data and Index

			tid = threadIdx.x;
			for (i = (data_size / 2); i > 0;) {
				if (tid < i) {
					//	if ((R_data[tid] >= R_data[tid + s]) && ((R_data[tid + s] < 0) && (R_data[tid] < 0))){
					//(R_data[tid + R_base] < 0) && (R_data[tid + R_base + i] < 0)&&
					if (R_data[tid + R_base] > R_data[tid + R_base + i]) { //is right-side value small?
						//if (R_data[tid + R_base + i] == -0.000000)
							 //R_data[tid + R_base + i] = 0.0;
						//if ((R_data[tid + R_base + i] <= -0.000001) || (R_data[tid + R_base + i] < 0) ) {	//only if the value on the right-side is -ive
						if (R_data[tid + R_base + i] <= -0.000001) {	//only if the value on the right-side is -ive
							R_data[tid + R_base] = R_data[tid + R_base + i];//put the smaller value to left-side
							R_index[tid + R_base] = R_index[tid + R_base + i];

							//notEntered = false;  //race condition avoided
							//notEntered = 0;  //race condition avoided
							int local_notEntered;
							local_notEntered = *(volatile int*) &notEntered;
							atomicCAS(&notEntered, local_notEntered, 0);
						}
					}
				}
				/*if (tid == 0)
				 printf("Data_size = %d ", i);*/
				__syncthreads();
				i >>= 1;
				if ((i != 1) && (i % 2) != 0) {	//if s is odd
					i = i + 1;
				}
			}
			// if (notEntered == false && tid == 2) { // tid==0 is always true if minValue is still -1 then what?
			if (threadIdx.x == 0) { // tid==0 is always true if minValue is still -1 then what?
			//	printf("\n Min Value = %f NewPivotCol = %d ", minValue,newpivotcol);
				if (notEntered == false) {
					minValue = R_data[R_base];
					newpivotcol = R_index[R_base];
					printf("\n Min Value = %f NewPivotCol = %d ", minValue,newpivotcol);
				}
			}
			__syncthreads(); //waiting for all threads to have same newpivotcol value

			//Debug----------
			/*if (threadIdx.x==0){
				printf("\n");
				for (int x=0;x<blockDim.x;x++){
					printf("(Data = %f Index = %d), ", R_data[R_base + x], R_index[R_base + x]);
				}
				printf("\n");
			}
			__syncthreads();*/
			//--------------
			//		}
			//		__syncthreads();	//here we have min and newpivotcol

			//Note to return minValue with Index (index of the simplex tableau and not the index of variable which will be -2)
			// ********* First Reduction Ends *************
			//  ******** Second Reduction Begins **********
			if (newpivotcol == -1) {//All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2; //No pivot column found so Optimal solution reached
				c=true; //can terminate
			} else { //if pivot column found then Find pivot row

				// ********** Second Reduction Process ******
				__shared__ float row_min;
				__shared__ int row_num;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
				if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					//if ((S_MAT[temp_index2] > 0) && (S_MAT[temp_index1] > 0)) {
					//Although Simplex algorithm says exclude zero quotient but case study show it takes zero as minimum positive ratio
					//if (S_MAT[temp_index2] > 0 && S_MAT[temp_index2] < INT_MAX) {
					if ((S_MAT[temp_index2] > 0) && (S_MAT[temp_index1] >= 0)) {
						//printf("\nS_MAT[temp_index2]= %f and S_MAT[temp_index1]= %f R_index= %d",S_MAT[temp_index2], S_MAT[temp_index1], k1);
						R_data[k1 + R_base] = (float)S_MAT[temp_index1] / (float)S_MAT[temp_index2]; //b_i / S_MAT[pivotcol]
						//-------------------------------------------------
						//Since there exists some feasible value which may be in the 1st location
						int local_notEntered2;
						local_notEntered2 = *(volatile int*) &notEntered2;
						atomicCAS(&notEntered2, local_notEntered2, 0);
						//-------------------------------------------------
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //Verified All data and index stored correctly with index as threadIdx.x
				//Debugging ---------------------------------------
/*				if(threadIdx.x==0){
					printf("Printing Last Row\n");
					for (int tt=0;tt<20;tt++)
						printf("%f ",R_data[tt + R_base]);
					printf("\nPrinting Done\n");
				}*/
				//----------------------------------------------------
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size
				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] > R_data[indexValue2 + s]) { //changed >= to > ToDo:: Accordingly Fix other bug
							R_data[indexValue2] = R_data[indexValue2 + s];	//For arranging in ascending order we better swap the value instead of only replacing
							R_index[indexValue2] = R_index[indexValue2 + s];
							//notEntered2 = false;

							//notEntered2 = 0;
/*	Fixing					int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);*/
						}
					}
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {	//if at least once swaped the flag 'notEntered2' will be equal to 0
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						printf("\nR_Data = %f pivotRow = %d", R_data[R_base], R_index[R_base]);
						//printf("\nR_Data next = %f R_Index next = %d", R_data[R_base + 1], R_index[R_base + 1]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
//					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						//if (notEntered2 == true) {
						//return -1;
						//printf("%f ", R_data[R_base]);
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						//}else {
						//return row_num;
						//printf("%f %d ", row_min, row_num);
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************
//*******************************************************************************************************
//			col = pivotCol;
			col = newpivotcol;
			//printf("Row= %d col = %d\n",row,col);
			if (row > -1) {	//some candidate leaving variable or pivot row have been found
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					/*if (tid == remember[tid - 2]) { //Before actual process convert back the Original column value of last row remembered in the previous iteration
						temp_index = Last_row + (tid * S_row) + base; //avoiding re-computation
						S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}*/

					if (remember[(tid - 2)] != 7777) { //Before actual process convert back the Original column value of last row remembered in the previous iteration
						printf("\nPreviously remembered column is %d ", remember[(tid - 2)]);
						//on the 2nd last row and on the remembered column replace back the original value
						temp_index = Last_row + (remember[(tid - 2)] * S_row) + base; //remember[(tid - 2)] gives the remembered column number
						S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
						remember[(tid - 2)] = 7777; //reset back to default value for next iterations
					}
				}		//Data Parallel section 1 done
				__syncthreads();
//*******************************************************************************************************
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();
//*******************************************************************************************************
				unsigned int temp_row_base = row + base;
				//S_MAT[temp_row_base + 1 * S_row] = S_MAT[temp_row_base + col * S_row]; //column 1 replaced with objective coefficient
				S_MAT[temp_row_base + 1 * S_row] = S_MAT[(S_row -1) + col * S_row + base]; //column 1 replaced by the new Last row containing objective coefficient
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[temp_row_base] = col - 1; //replacing entering variable index in leaving variable index (1-based indexing)
//*******************************************************************************************************
/*
				//Debugging after one iteration -----------------------
				if (threadIdx.x==0){
					printf("\n ----- Iteration Before update operation----- \n");
					for (int r=0;r<S_row;r++){
						for (int cl=0;cl<S_col;cl++){
							printf("%f  ",S_MAT[r+cl*S_row+base]);
						}
						printf("\n");
					}
					printf("\n");
				}
				//----------------------------------------------------- Correct replacement
*/
				tid = threadIdx.x;
				//Debug------------
				/*if (threadIdx.x==0)
					printf("\ncol1[row]=%f\n", col1[row]);*/
				//-----------------
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//updating pivot row by dividing, current pivot row value by (pivot element)
				}		//Data Parallel section 3 done
				__syncthreads();

				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < (S_row - 1)) { //updating all rows
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							float zeroTemp; 
							zeroTemp = col1[tid] * S_MAT[row + temp_index1];
							S_MAT[temp_index] = S_MAT[temp_index] - zeroTemp;
						} else {
							break;
						}
					}
				}	//Data Parallel section 4 done
				__syncthreads();

				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				//tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[(Last_row + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

				//Debugging after one iteration -----------------------
				/*if (threadIdx.x==0){
					printf("\n ----- Iteration ----- \n");
					for (int r=0;r<S_row;r++){
						for (int cl=0;cl<S_col;cl++){
							printf("%f  ",S_MAT[r+cl*S_row+base]);
						}
						printf("\n");
					}
					printf("\n");
				}*/
				//-----------------------------------------------------

			} else if (row == -1) { //No candidate leaving row have been found so remember this pivot column (and try next iteration although not mentioned in Algorithm)
				//This is actually the situation of UNBOUNDEDNESS but we are trying next iteration to see if the other candidate pivot column might give feasible pivot row
				//as it happened in the case of Helicopter model.
				//ToDo::Note we should have terminated this process when all columns have been tested. Otherwise for unbounded LPs our algorithm will not terminate.
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col; //remember this particular column and do not select this column in the next iterations
					printf("\n Remembered col = %d\n",col);
					rm++;
				}
				__syncthreads();
				//In the last row the value with the pivot column, col,
				temp_index = Last_row + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive so that this column will not be selected in the next iteration
				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				// tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[(Last_row + i * S_row) + base] < 0)) { //check if any negative in the last row (S_row -1)
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}/*
			//Debugging ------------------- Functioning without this as well
			else if (row == -2){
				//no entering variable (pivot column) found
				c=true;
			}
			// -------------------------------------*/
		} //end of while
		__syncthreads();
		
		if (threadIdx.x == 0) {
			Result[index] = S_MAT[(Last_row + (S_col - 1) * S_row) + base];
			//printf("\nResult Inside Kernel: %f \n",Result[index]);
		}
	}
}

__host__ Simplex::Simplex(unsigned int N_S) {
	number_of_LPs = N_S;
	M = 0;
	N = 0;
	c = 0;
	No_c = 0;
	R = (float*) calloc(N_S,sizeof(float));
	}

//get status of particular simplex
__host__ int Simplex::getStatus(int n) {
	int s;
	for (int i = 0; i < C.size1(); i++) {
		if (i == (n - 1)) {
			if (R[i] == -1) {
				s = 6;	// 6 = Simplex Is Unbounded
			} else if (R[i] > 0) {
				s = 2;	// 2= Simplex has feasible and Optimal solution
			}
		}
	}
	return s;

}	//get status of particular simplex

//get the No of simplex the object is ruuning on GPU
__host__ int Simplex::getNo_OF_Simplx() {
	return C.size1();
}	//get the No of simplex the object is ruuning on GPU

//get the result of all simplex
__host__ std::vector<float> Simplex::getResultAll() {

	std::vector<float> Res(C.size1());
	for (int i = 0; i < C.size1(); i++) {
		Res[i] = R[i];
	}
	return Res;
}

//get the result of all simplex

__host__ float Simplex::getResult(int n) {
	// get result of particular simplex
	float r;
	for (int i = 0; i < C.size1(); i++) {
		if (i == (n - 1)) {
			r = R[i];
		}
	}
	return r;
}	// get result of particular simplex

__host__ std::vector<int> Simplex::getStatusAll() {

	std::vector<int> Status(C.size1());
	for (int i = 0; i < C.size1(); i++) {
		if (R[i] == -1)
			Status[i] = 6;
		else
			Status[i] = 2;
	}
	return Status;
}	//get the status of all simplex

__host__ void Simplex::setConstratint(math::matrix<double> A, std::vector<double> B) {
	int N_S = number_of_LPs;
	orig_CoefficientMatrix = A;
	BoundValue = B;
	int No_O = A.size2();
	int No_C = A.size1();
	//M = No_C + 1;
	M = No_C + 2;//Extra row for coefficient of objective function
	N = No_O + 3 + No_C;//original variables + slack + 3 extra(index,pivot-col,b_i); artificial is not included now/here
	c = 1 + No_O;
//	MAT_COPY = (float *) calloc(N_S * M * N, sizeof(float));
	MAT = (float *) calloc(N_S * M * N, sizeof(float));

	/*
	 * Simplex tableau Re-Structure Amit :: Note The variables are implemented as 1-based indexing
	 * row-size= (m-constraints + 1-row Z and Optimal Solution value + 1-row for copy of coefficient of Objective function) = m+2
	 * column-size = (n-original-variables + m-slack-variables + a-artificial-variables+ 3 (1 for index, 1 for coefficient of the basic variables, 1 for bounds b_i's)
	 * column 0: index of basic/slack variables
	 * column 1: coefficient of basic variables
	 * column 2 to n: coefficients of variables (non-basic) --implemented as 2 to No_O+2 where No_O is the size of original variables
	 * column (n+1) to (n+m): includes slack variables --implemented as (No_O+2 +1) to (No_O+2+No_C) where No_C is the number of constraints comprising slack variables
	 * column (n+m+1) to (n+m+a): includes artificial variables --implemented after (No_O+2+No_C + 1) to <(N-1) where N is the total size of columns
	 * column last column (N-1) : bounds b_i
	 *
	 * row 0 through m: contains the coefficient for Simplex method algorithm
	 * row (m+1), the 2nd last row: contains the values of the operations (Cj - Zj) of each iterations starting from column 2 through (last - 1) and the last column contains the
	 * value of the optimal solution of each iterations.
	 * Last row (m+2): contains a copy of the coefficients of the objective function required for Simplex Algorithm. Column 2 through (N-1) is used to store these values
	 * ** NB: In phase-I : Artificial variables contains -1 and all values are 0. But in phase-II artificial variables are eliminated and the original coefficients are replaced
	 * 		  with original variables having their respective values and 0 for slack variables.
	 *
	 */
#pragma omp parallel for
	for (int s = 0; s < N_S; s++) {
		unsigned int some=M * N * s;
		for (int i = 0; i < (M - 2); i++) {
			for (int j = 0; j < N; j++) {
				if (j == 0) {	//index of basic/slack variables
					MAT[(int) ((i + j * M) + some)] = c + i;
				} else if (j > 1) { //excluding 'column 1' from loop
					if (j < (No_O + 2)) {	// coefficients of the variables (a.k.a. non-basic)
						MAT[(int) ((i + j * M) + some)] = (float) A(i,j - 2);
					} else if (j == (N - 1)) { //last column stores the bounds b_i
						MAT[(int) ((i + j * M) + some)] = (float) B[i];
					} else if (j < (N - 1)) { //includes slack variables
						MAT[(int) ((i + (No_O + 2 + i) * M) + some)] = 1;
					}
				}
			}
		}
	}

	/*  //Debugging
		printf("\n");
	for (int s = 0; s < N_S; s++) {
		unsigned int some=M * N * s;
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < N; j++) {
				std::cout<<MAT[(int)(i+ j*M +some)]<<"  ";
			}
			printf("\n");
		}
		printf("\n");
	}*/

}	

__host__ void Simplex::ComputeLP(math::matrix<float> &C1) {

	hipError_t err;
	unsigned int threads_per_block;	//Maximum threads depends on CC 1.x =512 2.x and > = 1024
	unsigned int number_of_blocks;//depends on our requirements (better to be much more than the number of SMs)
	int device;
	hipDeviceProp_t props;
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device);
	int No_C = orig_CoefficientMatrix.size1();
	C = math::matrix<float>(C1);
	int N_S = C.size1();
	int No_O = C.size2();
	M = No_C + 2, N = No_O + 3 + No_C;// M is now + 2 instead of +1

	int N_C = No_C;
	c = 1 + No_O;
	//int s;
#pragma omp parallel for
	for (int s = 0; s < N_S; s++) {
		unsigned int some = M * N * s; //base address for each LP
		for (int j = 2; j < (No_O+2); j++) { //Amit::Infact can be < (No_O+2) for Optimization
			//if (j < 2 + No_O) { //assigning objective coefficients of variables only
				//MAT[(int) (((M-1) + j * M) + some)] = -C(s, j - 2); //Last row (M-1)
			MAT[(int) (((M-2) + j * M) + some)] = C(s, j - 2); //Last row (M-1)  Amit::removed negative Now modified to M-2 the 2nd last row
			//Now keep a copy of coefficients of the objective function
			MAT[(int) (((M-1) + j * M) + some)] = C(s, j - 2); //slack is already zero as initialized
			//}
		}
	}

	//Debugging ---------------------------------------
	/*printf("\n******************* MAT tableau *******************\n");
	for (int s = 0; s < N_S; s++) {
		unsigned int some=M * N * s;
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < N; j++) {
				std::cout<<MAT[(int)(i+ j*M +some)]<<"  ";
			}
			printf("\n");
		}
		printf("\n");
	}*/
	// ---------------------------------------
	std::vector<int> rem;
	for (int i = 0; i < N_C; i++) {
		//std::cout<<BoundValue[i]<<"\n";
		if (BoundValue[i] < 0) {
			rem.push_back(i);
		}
	}

//	std::cout<<"Number of Artificial Variables = "<< rem.size()<<"\n";
	int nc = N + rem.size();
	
	threads_per_block = 32 * (nc / 32) + 32; //if count equal 0 than nc=N so works for all Model
	if (threads_per_block > props.maxThreadsPerBlock) //Assuming maximum threads supported by CC is 1024
		threads_per_block = props.maxThreadsPerBlock;

	int *R_index;	//reduction data
	float *R_data;	//reduction index
	err = hipMalloc((void **) &R_data, C1.size1() * threads_per_block * sizeof(float));//C1.size1() * 96 being the maximum threads
	err = hipMalloc((void **) &R_index, C1.size1() * threads_per_block * sizeof(int));//C1.size1() being the number of LPs
	err = hipMalloc((void **) &G_R, N_S * sizeof(float));//Doing it here for the First Time

	//printf("CUDA malloc R_index: %s\n", hipGetErrorString(err));
	//std::cout << "Number of threads per block = " << threads_per_block << "\n";

	if (rem.size() > 0) {
		//std::cout << "Simplex -Non-Basic Feasible Solution\n";
		N_MAT = (float *) calloc(N_S * M * nc, sizeof(float)); //initialized to zero (this tableau include artificial variables)

	/*
	 *Copied only RHS of all constraints to N_MAT from MAT
	 * This is done to copy b_i column from MAT into last column of N_MAT
	 */
#pragma omp parallel for
	for (int i = 0; i < N_S; i++) {
		int base = i * M * N;//base of every LP in MAT tableau
		int basen = i * M * nc;//base of every LP in N_MAT tableau (this include artificial variables)
		for (int j = 0; j < M; j++) {	//from every row/constraints
			//base=i*M*N;
			N_MAT[j + ((nc - 1) * M) + basen] = MAT[j + ((N - 1) * M) + base]; // N_MAT[lastCol] = MAT[lastCol]
		}
		for (int j = 2; j < (nc-1); j++) {	//from every column
			if (j<((No_O + 3 + No_C)-1))
				//N_MAT[(int) ((M-1) + j * M + some)] = MAT[(int) ((M - 1) + j * M + base)];//original and slack variables
				;//for Phase-I //original and slack variables will remain 0//N_MAT[(int) ((M-1) + j * M + some)] = 0;
			else
				N_MAT[(int) ((M-1) + j * M + basen)] = -1; //artificial variable
		}
	}

	//Debugging
/*	printf("\n******************* N_MAT tableau *******************\n");
	for (int s = 0; s < N_S; s++) {
		int basen = s * M * nc;//base of every LP in N_MAT tableau (this include artificial variables)
		for (int i = 0; i < M; i++) {	//from every row/constraints
			for (int j = 0; j < nc; j++) {	//from every column
				std::cout<<N_MAT[(i + (j * M) + basen)]<<" ";
			}
			printf("\n");
		}
		printf("\n");
	}*/
	// Verified upto here CPU code is fine


	//Creating Artificial Variables
#pragma omp parallel for
	for (int k = 0; k < N_S; k++) {
		bool once=false;
		int artif=0, ch;
		int base = k * M * N;//base of every LP in MAT tableau
		int basen = k * M * nc;//base of every LP in N_MAT tableau (this include artificial variables)
		for ( int i = 0; i < (M-1); i++) { //for every row including 2nd last row of the tableau :: leave last row
			ch = 0;
			for ( int j = 0; j < nc; j++) {  //for every column of the MAT and N_MAT tableau
				if (MAT[i + ((N - 1) * M) + base] < 0) { //this indicates the negative b_i from the MAT tableau
					if ((j >= (N - 1)) && (j < (nc - 1))) { //this indicate all columns that represent artificial variables
						if (!ch) {
							float v = N_MAT[(unsigned int)((i-1) + (j * M) + basen)]; //why (i-1)?
							if((once)&&(v==1)){ // computing v is meaningless since once is false and not made true anywhere so this will always be false
									N_MAT[(i + (j+1) * M) + basen] = 1;//so this block will never be executed ToDo:: can be skipped
							} else {
								N_MAT[(i + j * M) + basen] = 1;
							}
							ch = 1;	//this will allow populating 1, diagonally in artificial variables
						}
					} else if (j == (nc - 1)) { //this indicate the last column of N_MAT tableau which is b_i's
						N_MAT[(i + j * M) + basen] = -1 * N_MAT[(i + j * M) + basen]; //negating b_i's
					} else if (j == 1) { //the extra temporary working column
						N_MAT[(i + j * M) + basen] = -1; //why populate -1 only for negative b_i's in the extra column? May be used as coefficient of artificial variables
					} else if (j == 0) { //first index column
						//NOTE: Binayak used table index as variable indexing so it is 1-based Indexing
						//ToDo:: Amit detected Bugged here in index computation for Artificial variables
						//N_MAT[((i + j * M)) + basen] = (N + i)-2;//computes the index of artificial variables as n+m+a where size of vars, slacks and artificial are n,m and a respectively
						N_MAT[((i + j * M)) + basen] = (N + artif)-2;//increase index only when found artificial variable and not for every row i.
					//	std::cout<<" artif = " <<artif;
						artif++;//increase for next artificial variable found
						//std::cout<<" (N + i)-2 = " <<(N + i)-2;
					} else if (j > 1) { //negated all variables and slacks (only non-basic excluding artificial)
						N_MAT[(i + j * M) + basen] = -1 * (MAT[(i + j * M) + base]);
					}
				} else if ((i != (M - 2)) && (j < (N - 1))) { //except last row and last column of MAT i.e. b_i's
					N_MAT[(i + j * M) + basen] = MAT[(i + j * M) + base];//copy into N_MAT as it is
				} else if (i == (M - 2)) {
					if ((j >= (N - 1)) && (j < (nc - 1))) {
						N_MAT[(i + j * M) + basen] = -1; //ALL artificial variable coefficient is assigned -1
					}
				}
			}
		}
	}

	//Debugging
/*
	printf("\n******************* N_MAT tableau *******************\n");
	for (int s = 0; s < N_S; s++) {
		int basen = s * M * nc;//base of every LP in N_MAT tableau (this include artificial variables)
		for (int i = 0; i < M; i++) {	//from every row/constraints
			for (int j = 0; j < nc; j++) {	//from every column
				std::cout<<N_MAT[i + (j * M) + basen]<<" ";
			}
			printf("\n");
		}
		printf("\n");
	}
*/


//Creation of Last Row or Z-Value(Zj-Cj)
#pragma omp parallel for
	for (int k = 0; k < N_S; k++) {
		//int sum = 0;
		//base = k * M * N;
		int basen = k * M * nc;
		for (int k1 = 2; k1 < nc; k1++) {//for all columns upto b_i from column 2
			float sum = 0.0; //reset for every column k1 (objective function value)
			for (int j = 0; j < (M - 2); j++) { //for all rows except the 2nd last row for which this computation is performed and also last row
				sum = sum + (N_MAT[(j + k1 * M) + basen] * N_MAT[(j + 1 * M) + basen]); // column 1 currently contains -1, the coefficient of artificial variables
			}
			//std::cout << sum << "-"	<< N_MAT[((M - 1) + k1 * M) + basen];
			N_MAT[((M - 2) + k1 * M) + basen] = sum - N_MAT[((M - 2) + k1 * M) + basen]; //formula Zj - Cj
		}
	}
	//hipEvent_t start, stop;
	//Debugging ----------------------------
/*	std::cout << "\nSimplex AFTER CREATION OF Z before sending to GPU\n";
	for (int k = 0; k < N_S; k++) {
		int basen = k * M * nc;
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < nc; j++) {
				std::cout << N_MAT[(i + j * M) + basen] << "  ";
			}
			std::cout << "\n";
		}
		std::cout << "\n";
	}*/
	// ----------------------------

//		std::cout << "Before Kernel Called 1\n";
	hipMalloc((void **) &G_MAT, (N_S * M * nc * sizeof(float)));
	//printf("CUDA malloc G_MAT: %s\n", hipGetErrorString(err));
	hipMemcpy(G_MAT, N_MAT, (N_S * M * nc * sizeof(float)), hipMemcpyHostToDevice); //copy N_MAT from host to device in G_MAT
	//printf("CUDA malloc N_MAT: %s\n", hipGetErrorString(err));
	//	hipMemcpy(G_Sel, Sel, sizeof(int), hipMemcpyHostToDevice);
	//printf("CUDA malloc G_Sel: %s\n", hipGetErrorString(err));

	//mykernel<<<N_S, threads_per_block>>>(G_MAT, M, nc, G_R, N_S, G_Sel, R_data, R_index);
	mykernel<<<N_S, threads_per_block>>>(G_MAT, M, nc, G_R, N_S, R_data,R_index);
	hipDeviceSynchronize();
	hipMemcpy(R, G_R, N_S * sizeof(float), hipMemcpyDeviceToHost); //copy the reduced result in arrary R
	hipMemcpy(N_MAT, G_MAT, (N_S * M * nc * sizeof(float)),hipMemcpyDeviceToHost); //copy the current status of the G_MAT from device to N_MAT
	//for (int k = 0; k < N_S; k
/*	std::cout<< "\n***********Auxiliary SIMPLEX from GPU*************\n";
	for (int k = 0; k < N_S; k++) {
		// base=k*M*N;
		int basen = k * M * nc;
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < nc; j++) {
				std::cout << N_MAT[(i + j * M) + basen] << "  ";
			}
			std::cout << "\n";
		}
		std::cout << "\n";
	}*/
	//	std::cout << "Result for Artificial\n";
#pragma omp parallel for
		for (int i = 0; i < N_S; i++) {
			int base = i * M * N;
			int basen = i * M * nc;
			for (int j = 0; j < M; j++) { //for every row
				for (int k = 0; k < N; k++) { //for each column in MAT
					if (N_MAT[j + 0*M + basen] == (k + 1)) { //column 0 i.e. (row,0) has index of variables starting from (1 to n+m+a) so k starts from (1 to N, N=n+m+a+3)
						//So this condition will be met every value of k. if variable indexing is correctly assigned (Note Artificial is in-correct)
						N_MAT[j + 1*M + basen] = MAT[(M - 1) + (2 + k) * M + base]; //in column 1 of N_MAT replacing original problem's objective coefficients
					}
				}
			}
		}
/*	std::cout<< "\n***********Auxiliary SIMPLEX from GPU After modification: N_MAT *************\n";
	for (int k = 0; k < N_S; k++) {
		int basen = k * M * nc;
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < nc; j++) {
				std::cout << N_MAT[(i + j * M) + basen] << "  ";
			}
			std::cout << "\n";
		}
		std::cout << "\n";
	}*/

//std::cout<<"\nResult = "<<R[0]<<"\n";
#pragma omp parallel for
		for (int s = 0; s < N_S; s++) {
			if ((roundf(R[s]/10000)*10000) == 0) {
				//std::cout<<"\nInside ** IF*** Result = "<<R[0]<<"\n";
				//int sum = 0;
				int base = s * M * N;
				int basen = s * M * nc;
				for (int i = 0; i < N; i++) { //for each column i
					float sum = 0;
					for (int j = 0; j < (M - 1); j++) { //for every row j except the new last row of obj. coefficients
						if ((j < (M - 2))) { //except the last row
							if (i != (N - 1)) { //except the last column ie b_i's
								//std::cout<<N_MAT[(j+(i*M))+basen]<<"*"<<N_MAT[(j+(1*M))+basen]<<std::endl;
								sum = sum + (N_MAT[(j + (i * M)) + basen] * N_MAT[(j + (1 * M)) + basen]); //sum = sum + N_MAT[row,1] * N_MAT[row,i]
								MAT[(j + (i * M)) + base] = N_MAT[(j + (i * M)) + basen]; //copy data from N_MAT to MAT for every column i, as row by row (row is j)
							} else if (i == N - 1) { //for the last column ie b_i's
								sum = sum + (N_MAT[(j + (nc - 1) * M) + basen] * N_MAT[(j + (1 * M)) + basen]); //sum is for objective value column, the product of b_i's and coefficient's in column 1
								MAT[(j + (i * M)) + base] = N_MAT[(j + (nc - 1) * M) + basen]; //copy data of b_i's from N_MAT to MAT
							}
						}
						//if (j == (M - 1)) { //for the last row
						if (j == (M - 2)) { //for the 2nd last row
							if (i > 1) { // for all column from variables to slack variables excluding artificial variables
								//std::cout<<sum<<" And "<<MAT[(j+(i*M))+base]<<std::endl;
								//MAT[(j + (i * M)) + base] = MAT[(j + (i * M)) + base] + (-1) * sum; // Zj = Zj - Cj
								MAT[(j + (i * M)) + base] = sum + (-1 * MAT[(j + (i * M)) + base]); // Zj = Zj - Cj  Amit:: Corrected
							}
						}
					}
				}
			} else
				std::cout<<"The problem is Infeasible !!!\n";
		}
		hipFree (G_MAT);
		//		hipFree(G_R);
		//hipFree(G_Sel);
		//hipDeviceSynchronize();
		//		hipMalloc((void **) &G_R, N_S * sizeof(float));
		hipMalloc((void **) &G_MAT, (N_S * M * N * sizeof(float)));
		// printf("CUDA malloc G_MAT: %s\n", hipGetErrorString(err));
		hipMemcpy(G_MAT, MAT, (N_S * M * N * sizeof(float)), hipMemcpyHostToDevice); //Now copy MAT to device G_MAT
		//printf("CUDA malloc N_MAT: %s\n", hipGetErrorString(err));
		//	hipMemcpy(G_Sel, Sel, sizeof(int), hipMemcpyHostToDevice);
		//printf("CUDA malloc G_Sel: %s\n", hipGetErrorString(err));
//std::cout<<"Kernel Called 2\n";
	//Debugging
/*
		std::cout<< "\n***********Before SIMPLEX sent to GPU: MAT *************\n";
		for (int k = 0; k < N_S; k++) {
			int basen = k * M * N;
			for (int i = 0; i < M; i++) {
				for (int j = 0; j < N; j++) {
					std::cout << MAT[(int)(i + (j * M) + basen)] << "  ";
				}
				std::cout << "\n";
			}
			std::cout << "\n";
		}
*/

		mykernel<<<N_S, threads_per_block>>> (G_MAT, M, N, G_R, N_S, R_data, R_index);
		hipDeviceSynchronize();
		hipMemcpy(R, G_R, N_S * sizeof(float), hipMemcpyDeviceToHost); //store the result in arrary R

		hipMemcpy(MAT, G_MAT, (N_S * M * N * sizeof(float)), hipMemcpyDeviceToHost);
		std::cout<< "\n***********Final SIMPLEX from GPU*************\n";
		for (int k = 0; k < N_S; k++) {
			int basen = k * M * N;
			for (int i = 0; i < M; i++) {
				for (int j = 0; j < N; j++) {
					std::cout << MAT[(int)(i + (j * M) + basen)] << "  ";
				}
				std::cout << "\n";
			}
			std::cout << "\n";
		}

		//std::cout
		// << "***********Final SIMPLEX from GPU*************\n Time took:\n";*/
	}
	//hipFree(G_MAT);
	//hipFree(G_Sel);
	//hipFree(G_R);
	hipFree(R_index);	//Only to synchronize with the cudamemcpy
	//hipFree(R_data);	//Only to synchronize with the cudamemcpy
}

