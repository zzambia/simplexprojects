#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <stdio.h>
#include <stdlib.h>
#include "simplex.cuh"
#include "math/glpk_lp_solver/glpk_lp_solver.h"
#include <vector>
#include "math/matrix.h"
#include <climits>
#include <iostream>
#include "boost/timer/timer.hpp"
#include "sys/time.h"

int main(int argc, char *argv[]) {
	unsigned int LP_size = 1, avg, dimension, art, stream;
	math::matrix<double> A;
	math::matrix<float> C, newC;
	std::vector<double> b;
	std::vector<float> result;
	std::vector<int> status_val;
	double Final_Time;

	if (argc > 1) {
		if (argc != 5) {		//1(ApplicationName) + 4 (Input Arguments)
			std::cout << "\nInsufficient Number of Arguments!!!\n";
			std::cout << "Correct Usages/Syntax:\n";
			std::cout
					<< "./ProjName --'Dimension'--'Average'--'Batch-size' ---'Artificial-Variables'!!\n";
			std::cout
					<< "argument 1) Dimension -- select the dimension of LP to be solved\n";
			std::cout
					<< "argument 2) Average -- select the number of runs for average readings\n";
			std::cout
					<< "argument 3) Batch-size -- select the number of LPs to be solved\n";
			std::cout
					<< "argument 4) Artificial-Variables -- Number of negative on RHS of the LP constraints\n";
			//std::cout<< "argument 5) Streams -- 0 for no streaming, 1 or 10 or 'n' to select the number of streams \n";
			return 0;
		} else {
			unsigned int num;
			dimension = atoi(argv[1]);
			num = atoi(argv[2]);
			avg = num;
			LP_size = atoi(argv[3]);
			art = atoi(argv[4]);
			//stream = atoi(argv[5]);
		}
	}
	glpk_lp_solver lp;
	unsigned int status = 0, res1 = 0, res2 = 0;
	//**** Creating Random Lp's Any x Any ****
	int sign = -1, sum1;

	while (status != 5) {
		A.resize(dimension, dimension);
		b.resize(dimension);
		for (unsigned int j = 0; j < dimension; j++) {
			for (unsigned int k = 0; k < dimension; k++) {
				A(j, k) = rand() % (k + 10) + 1;
			}

			if (j < art) {
				res1 = (rand() % 50);
				res2 = (res1 + 1);
				sum1 = res2 * sign;
				b[j] = (double) sum1;
				A(j, j) = A(j, j) * sign;
			} else
				b[j] = (rand() % 50) + 1;
		}
		//** Setting current Lp to GLPK
		lp.setMin_Or_Max(2);
		lp.setConstraints(A, b, 1);
		status = lp.TestConstraints();
	}
	/*
	 * Experimental Note:
	 *  We generate randomly the first LP problem (the matrix A and vector b) and then generate
	 *  randomly the objective function(s). For the ease generation process we take all the
	 *  LP problems to be the same LP generated above. However, for each of this LPs we generate
	 *  randomly different objective functions.
	 *
	 *  Also, to record the average time we ignore the first reading taken in GPU as it does not
	 *  reflect the correct computation time because for the first GPU call it also include an
	 *  extra overhead of GPU initialization time.
	 */


	double sum = 0.0;
	double wall_clock, user_clock, system_clock, return_Time;
	boost::timer::cpu_timer tt1, tt2;	//tt1 -- Variable declaration

	C.resize(LP_size, dimension);	//objective function
	for (unsigned int i = 0; i < C.size1(); i++) {
		for (unsigned int j = 0; j < C.size2(); j++) {
			C(i, j) = rand() % (j + 1) + 1;
		}
	}

	std::cout << "\n*****GPU RESULT*****\n";

	for (unsigned int i = 0; i <= avg; i++) {
		Simplex s(C.size1());
		s.setConstratint(A, b);	//setting constraints also recorded like in GLPK for independent LP
		tt1.start();
		s.ComputeLP(C);
		tt1.stop();
		if (i == 0) {
			wall_clock = tt1.elapsed().wall / 1000000; //convert nanoseconds to milliseconds
			return_Time = wall_clock / (double) 1000; //convert milliseconds to seconds
			std::cout << "Iter = " << i << " Time = " << return_Time
					<< std::endl;
		} else {
			wall_clock = tt1.elapsed().wall / 1000000; //convert nanoseconds to milliseconds
			return_Time = wall_clock / (double) 1000; //convert milliseconds to seconds
			std::cout << "Iter = " << i << " Time = " << return_Time
					<< std::endl;
			sum = sum + return_Time; //convert nanoseconds to milliseconds
		}
		result = s.getResultAll();
	}
	Final_Time = sum / avg;
	std::cout << "\nNumber of Simplex Solved = " << C.size1() << std::endl;
	std::cout << "\nBoost Time taken:Wall  (in Seconds):: GPU= "
			<< (double) Final_Time << std::endl;
	std::cout << "\n**Answer_Of_All_Simplex**\n";

	//Computation for CPU ie GLPK
	//	sum=0.0;
	std::cout << "\n*****GLPK RESULT*****\n";
	std::vector<double> dir(dimension);

	//***** MODEL SELECTION *****
	double res = 0.0;
	double batchTime = 0.0, AvgBatchTime = 0.0;
	std::vector<double> resul(C.size1());
	for (int i = 1; i <= avg; i++) {
		//batchTime = 0.0;
		tt1.start();
		for (int i = 0; i < C.size1(); i++) {
			glpk_lp_solver mylp;
			mylp.setMin_Or_Max(2);
			for (int j = 0; j < dimension; j++) {
				dir[j] = C(i, j);
			}
			mylp.setConstraints(A, b, 1); //this function actually determines independent LP in GLPK
			res = mylp.Compute_LLP(dir); //We consider every dir an independent LP problem
			resul[i] = res;
		}
		tt1.stop();
		wall_clock = tt1.elapsed().wall / 1000000; //convert nanoseconds to milliseconds
		return_Time = wall_clock / (double) 1000; //convert milliseconds to seconds
		batchTime = return_Time; //convert nanoseconds to milliseconds
		AvgBatchTime = AvgBatchTime + batchTime;
	}
	Final_Time = AvgBatchTime / avg;
	std::cout << "\nNumber of Simplex Solved = " << C.size1() << std::endl;
	std::cout << "\nBoost Time taken:Wall  (in Seconds):: GLPK= "
			<< (double) Final_Time << std::endl;

	int max = 5;	//Verifying results of only first 5 LPs.
	if (LP_size < max)
		max = LP_size;

	std::cout << "\nVERIFICATION FOR CORRECTNESS\n";
	for (int i = 0; i < max; i++) {
		std::cout << "GLPK: " << resul[i] << " || GPU: " << result[i] << std::endl;
	}

	return 0;
}

