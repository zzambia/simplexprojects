#include "hip/hip_runtime.h"
#include "simplex.cuh"
#include<omp.h>
#include "iostream"
#include <math.h>

//Method : To determine Pivot Column i.e., the entering variable we use the most negative value approach
//Race Condition completely avoided
//Both Reduction implemented- One for finding Pivot column and the other for finding Pivot Row
//Implemented Reduction. But without Streams.


// 2nd Method:: Random negative value
__global__ void mykernel2(float *S_MAT, int S_row, int S_col, float *Result,
		int S_N, float *R_data, int *R_index) {
	//int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int index = blockIdx.x;
	if (index < S_N) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		__shared__ float col1[1024];	//pivotColumn
		/*************/
		//if (threadIdx.x == 0) {
			c = false;
			rm = 0;
			row = -1;		//pivotRow
			pivotCol = -1;
		//}
		__syncthreads();
		while (!c) {
			//__syncthreads();
			int Last_row = S_row - 1;
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
				minValue = 0;
				newpivotcol = -1;
				c = true;
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			//for (int j = 2; j < S_col - 1; j++) {//only last row but all column
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) {
				int j = threadIdx.x;
				unsigned int temp_index1 = Last_row + j * S_row + base; //avoiding re-computation
				if (S_MAT[temp_index1] < minValue) {
					//minValue = S_MAT[temp_index1];
					newpivotcol = j; //"Any(Random) negative coefficient rule"
					/*
					int local_NewPivotCol;
					local_NewPivotCol = *(volatile int*) &newpivotcol;
					atomicCAS(&newpivotcol, local_NewPivotCol, j);
					*/
					/*
					 http://stackoverflow.com/questions/27616417/cuda-is-there-any-way-to-prevent-other-threads-from-changing-a-shared-or-global
					 if (atomicCAS(&newpivotcol, local_NewPivotCol, j)==local_NewPivotCol){
					 //this thread won the write
					 printf("Thread ID = %d ",threadIdx.x);
					 }*/
					//break;
				}
			}
			__syncthreads(); //here we have min and newpivotcol

			//  ******** Second Reduction Begins **********

			if (newpivotcol == -1) { //All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2;
			} else {
				// ********** Second Reduction Process ******
				//in order to avoid global memory transfer:: Using the same R_data and R_index global memory
				__shared__ float row_min;
				__shared__ int row_num;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
				if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					if ((S_MAT[temp_index2] > 0) && (S_MAT[temp_index1] > 0)) {
						R_data[k1 + R_base] = S_MAT[temp_index1]
								/ S_MAT[temp_index2];
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //here have all values from 0 to BLOCK_SIZE
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size
				/*if (threadIdx.x == 0) {
				 printf("\nR_data \n");
				 for (int x = 0; x < Last_row; x++)
				 printf("%f  ", R_data[x]);
				 printf("\nR_Index \n");
				 for (int x = 0; x < Last_row; x++)
				 printf("%d  ", R_index[x]);
				 printf("Data_size2 = %d ", data_size2);
				 }
				 __syncthreads();*/
				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] >= R_data[indexValue2 + s]) {
							R_data[indexValue2] = R_data[indexValue2 + s];
							R_index[indexValue2] = R_index[indexValue2 + s];
							//notEntered2 = false;
							notEntered2 = 0;
							/*int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);
							*/
						}
					}
//					if (tid == 0)
//						printf("Data_size = %d ", s);
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						//printf("R_Data = %f R_Index = %d", R_data[R_base], R_index[R_base]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						//if (notEntered2 == true) {
						//return -1;
						//printf("%f ", R_data[R_base]);
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						//}else {
						//return row_num;
						//printf("%f %d ", row_min, row_num);
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************

			//col = S_Sel[index];
			//col = *S_Sel;
			col = pivotCol;
			if (row > -1) {
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					if (tid == remember[tid - 2]) {
						temp_index = (S_row - 1) + (tid * S_row) + base; //avoiding re-computation
						S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}
				}		//Data Parallel section 1 done
				__syncthreads();
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();

				unsigned int temp_row_base = row + base;//avoiding re-computation
				S_MAT[temp_row_base + S_row] =
						S_MAT[temp_row_base + col * S_row];
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[row + base] = col - 1;//now temp_row_base is not required
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = S_MAT[temp_index] / S_MAT[row_base + S_row];
				}		//Data Parallel section 3 done
				__syncthreads();
				//printf("Row here = %d",row);
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							S_MAT[temp_index] = S_MAT[temp_index]
									- (col1[tid] * S_MAT[row + temp_index1]);
						} else {
							break;
						}
					}
				}	//Data Parallel section 4 done
				__syncthreads();

				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				//tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[((S_row - 1) + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

			} else if (row == -1) {
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col;
					rm++;
				}
				__syncthreads();

				temp_index = (S_row - 1) + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive
				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				// tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[((S_row - 1) + i * S_row) + base] < 0)) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}
		} //end of while
		__syncthreads();
		if (threadIdx.x == 0) {
			//printf("Value = %f ",S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base]);
			Result[index] = S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base];
		}
	}
}


//1st Method : Most Negative Value approach
__global__ void mykernel(float *S_MAT, int S_row, int S_col, float *Result,
		int S_N, float *R_data, int *R_index) {
	//int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int index = blockIdx.x;
	if (index < S_N) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		__shared__ float col1[1024];	//pivotColumn
		/*************/
		if (threadIdx.x == 0) {
			c = false;
			rm = 0;
			row = -1;		//pivotRow
			pivotCol = -1;
		}
		__syncthreads();
		while (!c) {
			__syncthreads();
		/*if(threadIdx.x==0){
			for(int ik=0;ik<S_row;ik++){
					for(int jk=0;jk<S_col;jk++){
						printf("%f  ",S_MAT[ik+jk*S_row+index]);					
					}		
					printf("\n");	
				}
			printf("\n\n\n\n");
			printf("Row= %d col = %d\n",row,pivotCol);
			printf("\n\n\n\n");
						
		}
		__syncthreads(); */
			int Last_row = S_row - 1;
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			__shared__ int notEntered;
			__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
				minValue = 0;
				newpivotcol = -1;
				notEntered = 1;
				c = true;
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			// Since keeping limit only upto (S_col - 1) which is not equal to BLOCK_SIZE creates problem
			// in using syncthreads() inside Reduction for-loop so use all threads(all R_data)
			//int data_size = (S_col - 1) - 2;
			int data_size = blockDim.x;
			tid = threadIdx.x;
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) {
				//tid = threadIdx.x - 2;//here my tid should be from 0 to (evenThreadId - 1)
				//int j = threadIdx.x;//is the actual column/index number less than (S_col - 1)
				//printf("Data_size = %d ", data_size);
				temp_index = Last_row + tid * S_row + base;	//avoiding re-computation
				R_data[tid + R_base] = S_MAT[temp_index];	//	g_data[i];
				R_index[tid + R_base] = tid;//tid; should be the real index of the data
			} else {
				R_data[tid + R_base] = INT_MAX;	//	g_data[i];
				R_index[tid + R_base] = tid;	//tid;
			}
			__syncthreads();//here will have all values in shared memory from 0 to BLOCK_SIZE

			tid = threadIdx.x;
			for (i = (data_size / 2); i > 0;) {
				if (tid < i) {
					//	if ((R_data[tid] >= R_data[tid + s]) && ((R_data[tid + s] < 0) && (R_data[tid] < 0))){
					//(R_data[tid + R_base] < 0) && (R_data[tid + R_base + i] < 0)&&
					if (R_data[tid + R_base] > R_data[tid + R_base + i]) { //is right-side value small?
						
						//if (R_data[tid + R_base + i] == -0.000000)
							 //R_data[tid + R_base + i] = 0.0;
						if (R_data[tid + R_base + i] <= -0.000001 ) {	//only if the value on the right-side is -ive
							R_data[tid + R_base] = R_data[tid + R_base + i];//put the smaller value to left-side
							R_index[tid + R_base] = R_index[tid + R_base + i];

							//notEntered = false;  //race condition avoided

							//notEntered = 0;  //race condition avoided
							int local_notEntered;
							local_notEntered = *(volatile int*) &notEntered;
							atomicCAS(&notEntered, local_notEntered, 0);

						}
					    
					}
				}
				/*if (tid == 0)
				 printf("Data_size = %d ", i);*/
				__syncthreads();
				i >>= 1;
				if ((i != 1) && (i % 2) != 0) {	//if s is odd
					i = i + 1;
				}
			}
			// if (notEntered == false && tid == 2) { // tid==0 is always true if minValue is still -1 then what?
			if (threadIdx.x == 0) { // tid==0 is always true if minValue is still -1 then what?
				if (notEntered == false) {
					minValue = R_data[R_base];
					newpivotcol = R_index[R_base];
					//printf("\nminValue = %f newpivotcol = %d ", minValue,newpivotcol);
				}
			}
			__syncthreads(); //waiting for all threads to have same newpivotcol value
			//		}
			//		__syncthreads();	//here we have min and newpivotcol
			// ********* First Reduction Ends *************
			//  ******** Second Reduction Begins **********
			/*			if (threadIdx.x == 0) {
			 if (newpivotcol == -1) {
			 //return -2;
			 row = -2;
			 } else {
			 float row_min = INT_MAX;
			 float row_num = -1;
			 //TODO:: this temp_res can be an array of value computed in parallel
			 //TODO:: row_min and row_num can then be computed using reduction
			 for (i = 0; i < S_row - 1; i++) {

			 temp_index = newpivotcol * S_row + i + base; //avoiding re-computation
			 temp_index1 = i + (S_col - 1) * S_row + base; //avoiding re-computation
			 if ((S_MAT[temp_index] > 0)
			 && (S_MAT[temp_index1] > 0)) {
			 float temp_res = S_MAT[temp_index1]
			 / S_MAT[temp_index]; //avoiding re-computation
			 if (temp_res <= row_min) {
			 row_min = temp_res;
			 row_num = i;
			 }
			 }
			 }
			 // *S_Sel = newpivotcol;
			 pivotCol = newpivotcol;
			 //S_Sel[index] = newpivotcol;
			 if (row_min == INT_MAX) {
			 //return -1;
			 row = -1;
			 }
			 if (row_num != -1) {
			 //return row_num;
			 row = row_num;
			 }
			 }
			 } //end of one thread
			 __syncthreads();*/
			if (newpivotcol == -1) {//All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2;
			} else {
				// ********** Second Reduction Process ******
				//in order to avoid global memory transfer:: Using the same R_data and R_index global memory
				__shared__ float row_min;
				__shared__ int row_num;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
				if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					if ((S_MAT[temp_index2] > 0) && (S_MAT[temp_index1] > 0)) {
						R_data[k1 + R_base] = S_MAT[temp_index1]/ S_MAT[temp_index2];
						//R_data[k1 + R_base] = roundf(((S_MAT[temp_index1]/ S_MAT[temp_index2])/1000000)*1000000);
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //here have all values from 0 to BLOCK_SIZE
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size
				/*if (threadIdx.x == 0) {
				 printf("\nR_data \n");
				 for (int x = 0; x < Last_row; x++)
				 printf("%f  ", R_data[x]);
				 printf("\nR_Index \n");
				 for (int x = 0; x < Last_row; x++)
				 printf("%d  ", R_index[x]);
				 printf("Data_size2 = %d ", data_size2);
				 }
				 __syncthreads();*/
				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] >= R_data[indexValue2 + s]) {
							R_data[indexValue2] = R_data[indexValue2 + s];
							R_index[indexValue2] = R_index[indexValue2 + s];
							//notEntered2 = false;

							//notEntered2 = 0;
							int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);
						}
					}
//					if (tid == 0)
//						printf("Data_size = %d ", s);
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						//printf("R_Data = %f R_Index = %d", R_data[R_base], R_index[R_base]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						//if (notEntered2 == true) {
						//return -1;
						//printf("%f ", R_data[R_base]);
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						//}else {
						//return row_num;
						//printf("%f %d ", row_min, row_num);
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************

			//col = S_Sel[index];
			//col = *S_Sel;
			col = pivotCol;
			//printf("Row= %d col = %d\n",row,col);
			if (row > -1) {
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					if (tid == remember[tid - 2]) {
						temp_index = (S_row - 1) + (tid * S_row) + base; //avoiding re-computation
						//if (S_MAT[temp_index] == -0.0)
							//S_MAT[temp_index] = -1 * 0.0; //replacing back to original
						//else
							S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}
				}		//Data Parallel section 1 done
				__syncthreads();
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();

				unsigned int temp_row_base = row + base;//avoiding re-computation
				S_MAT[temp_row_base + S_row] =
						S_MAT[temp_row_base + col * S_row];
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[row + base] = col - 1;//now temp_row_base is not required
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = roundf(((S_MAT[temp_index] / col1[row])/1000000)*1000000);//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = S_MAT[temp_index] / S_MAT[row_base + S_row];
				}		//Data Parallel section 3 done
				__syncthreads();
				//printf("Row here = %d",row);
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							float zeroTemp; 
							zeroTemp = col1[tid] * S_MAT[row + temp_index1];
							S_MAT[temp_index] = S_MAT[temp_index] - zeroTemp;
							//S_MAT[temp_index] = roundf(((S_MAT[temp_index] - zeroTemp)/1000000)*1000000);
							//if (S_MAT[temp_index] == -0.0)
								//S_MAT[temp_index]= 0.0;
							
						} else {
							break;
						}
					}
				}	//Data Parallel section 4 done
				__syncthreads();

				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				//tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[((S_row - 1) + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

			} else if (row == -1) {
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col;
					rm++;
				}
				__syncthreads();

				temp_index = (S_row - 1) + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				//if (S_MAT[temp_index] == -0.0)
					//S_MAT[temp_index] = 0;	//remembering by making positive
				//else
					S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive
				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				// tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[((S_row - 1) + i * S_row) + base] < 0)) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}
		
		} //end of while
		__syncthreads();
		
		if (threadIdx.x == 0) {
	/*
				if(threadIdx.x==0){
			for(int ik=0;ik<S_row;ik++){
					for(int jk=0;jk<S_col;jk++){
						printf("%f  ",S_MAT[ik+jk*S_row+index]);					
					}		
					printf("\n");	
				}
			printf("\n\n\n\n");
			printf("Row= %d col = %d\n",row,pivotCol);
			printf("\n\n\n\n");
						
		}
		__syncthreads();
			//printf("Value = %f ",S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base]);
			
*/			Result[index] = S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base];
			//printf("\nResult Inside Kernel: %f \n",Result[index]);
		}
	}
}

__host__ Simplex::Simplex(unsigned int N_S) {
	number_of_LPs = N_S;
	M = 0;
	N = 0;
	c = 0;
	No_c = 0;
	R = (float*) calloc(N_S,sizeof(float));
	}

//get status of particular simplex
__host__ int Simplex::getStatus(int n) {
	int s;
	for (int i = 0; i < C.size1(); i++) {
		if (i == (n - 1)) {
			if (R[i] == -1) {
				s = 6;	// 6 = Simplex Is Unbounded
			} else if (R[i] > 0) {
				s = 2;	// 2= Simplex has feasible and Optimal solution
			}
		}
	}
	return s;

}	//get status of particular simplex

//get the No of simplex the object is ruuning on GPU
__host__ int Simplex::getNo_OF_Simplx() {
	return C.size1();
}	//get the No of simplex the object is ruuning on GPU

//get the result of all simplex
__host__ std::vector<float> Simplex::getResultAll() {

	std::vector<float> Res(C.size1());
	for (int i = 0; i < C.size1(); i++) {
		Res[i] = R[i];
	}
	return Res;
}

//get the result of all simplex

__host__ float Simplex::getResult(int n) {
	// get result of particular simplex
	float r;
	for (int i = 0; i < C.size1(); i++) {
		if (i == (n - 1)) {
			r = R[i];
		}
	}
	return r;
}	// get result of particular simplex

__host__ std::vector<int> Simplex::getStatusAll() {

	std::vector<int> Status(C.size1());
	for (int i = 0; i < C.size1(); i++) {
		if (R[i] == -1)
			Status[i] = 6;
		else
			Status[i] = 2;
	}
	return Status;
}	//get the status of all simplex

__host__ void Simplex::setConstratint(math::matrix<double> A,
		std::vector<double> B) {
	int N_S = number_of_LPs;
	orig_CoefficientMatrix = A;
	BoundValue = B;
	int No_O = A.size2();
	int No_C = A.size1();
	M = No_C + 1;
	N = No_O + 3 + No_C;
	c = 1 + No_O;
//	MAT_COPY = (float *) calloc(N_S * M * N, sizeof(float));
	MAT = (float *) calloc(N_S * M * N, sizeof(float));
int s;
#pragma omp parallel for
	for (s = 0; s < N_S; s++) {
		unsigned int some=M * N * s;
		for (int i = 0; i < M - 1; i++) {
			for (int j = 0; j < N; j++) {
				if (j == 0) {
					MAT[(int) ((i + j * M) + some)] = c + i;
					//MAT_COPY[(int) ((i + j * M) + some)] = c + i;
				} else if (j > 1) {
					if (j < (No_O + 2)) {
						//MAT_COPY[(int) ((i + j * M) + some)] = (float) A(i,j - 2);
						MAT[(int) ((i + j * M) + some)] = (float) A(i,j - 2);
					} else if (j == N - 1) {
						//MAT_COPY[(int) ((i + j * M) + some)] = (float) B[i];
						MAT[(int) ((i + j * M) + some)] = (float) B[i];
					} else if (j < N - 1) {
						MAT[(int) ((i + (No_O + 2 + i) * M) + some)] = 1;
						//MAT_COPY[(int) ((i + (No_O + 2 + i) * M) + some)] = 1;
					}
				}
			}
		}
	}
}	

__host__ void Simplex::ComputeLP(math::matrix<float> &C1) {

	hipError_t err;
	unsigned int threads_per_block;	//Maximum threads depends on CC 1.x =512 2.x and > = 1024
	unsigned int number_of_blocks;//depends on our requirements (better to be much more than the number of SMs)
	int device;
	hipDeviceProp_t props;
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device);
	int No_C = orig_CoefficientMatrix.size1();
	C = math::matrix<float>(C1);
	int N_S = C.size1();
	int No_O = C.size2();
	M = No_C + 1, N = No_O + 3 + No_C;
	int N_C = No_C;
	c = 1 + No_O;
int s;
#pragma omp parallel for
	for (s = 0; s < N_S; s++) {
		unsigned int some = M * N * s;
		for (int i = M - 1; i < M; i++) {
			for (int j = 2; j < N; j++) {
				if (j < 2 + No_O) {
					MAT[(int) ((i + j * M) + some)] = -C(s, j - 2);
					//MAT_COPY[(int) ((i + j * M) + some)] = -C(s, j - 2);//Added for New Copy
				}	
			}
		}
	}
	std::vector<int> rem;
	for (int i = 0; i < N_C; i++) {
		
		//std::cout<<BoundValue[i]<<"\n";
		if (BoundValue[i] < 0) {
			rem.push_back(i);
			
		}
	}

	//std::cout<<"C= "<< rem.size()<<"\n";
	int nc = N + rem.size();
	
	threads_per_block = 32 * (nc / 32) + 32; //if count equal 0 than nc=N so works for for Model
	if (threads_per_block > props.maxThreadsPerBlock) //Assuming maximum threads supported by CC is 1024
		threads_per_block = props.maxThreadsPerBlock;

	int *R_index;	//reduction data
	float *R_data;	//reduction index
	err = hipMalloc((void **) &R_data,
			C1.size1() * threads_per_block * sizeof(float));//C1.size1() * 96 being the maximum threads
	err = hipMalloc((void **) &R_index,
			C1.size1() * threads_per_block * sizeof(int));//C1.size1() being the number of LPs

	err = hipMalloc((void **) &G_R, N_S * sizeof(float));//Doing it here for the First Time

	//printf("CUDA malloc R_index: %s\n", hipGetErrorString(err));
	//std::cout << "Number of threads per block = " << threads_per_block << "\n";

	if (rem.size() > 0) {
	//int s;
/* Now MAT_COPY not required
 * 	#pragma omp parallel for
	for (s = 0; s < N_S; s++) {
		unsigned int some=M * N * s;
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < N; j++) {
					MAT[(int) ((i + j * M) + some)] = MAT_COPY[(int) ((i + j * M) + some)];
			}
		}
	}*/

	//Helicopter model has no negative bound so count=0
	//std::cout << "Simplex -Non_ Basic Feasible Solution\n";		
	N_MAT = (float *) calloc(N_S * M * nc, sizeof(float));
		int i;

//Copied only RHS of all constraints to N_MAT from MAT
#pragma omp parallel for
		for (i = 0; i < N_S; i++) {
			for (int j = 0; j < M; j++) {
				int base = i * M * N;
				int basen = i * M * nc;
				//base=i*M*N;
				N_MAT[j + ((nc - 1) * M) + basen] = MAT[j + ((N - 1) * M) + base];
			}
		}
//Creating Artificial Variables	
int k,ch,base,basen;
bool once=false;
#pragma omp parallel for
for (k = 0; k < N_S; k++) {
				base = k * M * N;
				basen = k * M * nc;
				for ( int i = 0; i < M; i++) {
					ch = 0;
					for ( int j = 0; j < nc; j++) {
						if (MAT[i + ((N - 1) * M) + base] < 0) {
							if ((j >= (N - 1)) && (j < (nc - 1))) {
								if (!ch) {
									float v=N_MAT[(unsigned int)((i-1) + (j * M) + basen)];
									//std::cout<<"V  = " <<v <<"\n";
									if((once)&&(v==1)){
											//std::cout<<"V inside = " <<v <<"\n";
											N_MAT[(i + (j+1) * M) + basen] = 1;
										}
									else{
										N_MAT[(i + j * M) + basen] = 1;
									    
										}
									ch = 1;
								}
							} else if (j == (nc - 1)) {
								N_MAT[(i + j * M) + basen] = -1
										* N_MAT[(i + j * M) + basen];
							} else if (j == 1) {
								N_MAT[(i + j * M) + basen] = -1;
							} else if (j == 0) {
								//N_MAT[((i + j * M)) + basen] = ((N - M) + i + 3+ No_O);
								N_MAT[((i + j * M)) + basen] = (N + i)-2;
							} else if (j > 1) {
								N_MAT[(i + j * M) + basen] = -1
										* (MAT[(i + j * M) + base]);
							}
						} else if ((i != (M - 1)) && (j < N - 1)) {
							N_MAT[(i + j * M) + basen] =
									MAT[(i + j * M) + base];
						} else if ((i != (M - 1)) && (j == nc - 1)) {
							continue;
						} else if (i == (M - 1)) {
							if ((j >= (N - 1)) && (j < nc - 1)) {
								N_MAT[(i + j * M) + basen] = -1;
							} else if (j == nc - 1) {
								continue;
							} 
						}
					}
				}
			}	

//Creation of Last Row or Z-Value(Z-C)
	#pragma omp parallel for
		for (k = 0; k < N_S; k++) {
			//int sum = 0;
			//base = k * M * N;
			int basen = k * M * nc;
			for (int k1 = 2; k1 < nc; k1++) {
				float sum = 0.0;
				for (int j = 0; j < (M - 1); j++) {
					sum = sum + (N_MAT[(j + k1 * M) + basen] * N_MAT[(j + 1 * M) + basen]);
					//std::cout << N_MAT[(j + k1 * M) + basen] << "*"<< N_MAT[(j + 1 * M) + basen] << "\t";
				}
				//std::cout << sum << "-"	<< N_MAT[((M - 1) + k1 * M) + basen];
				N_MAT[((M - 1) + k1 * M) + basen] = sum - N_MAT[((M - 1) + k1 * M) + basen];
				//std::cout << "=" << N_MAT[((M - 1) + k1 * M) + basen]
				//<< "\n";
			}
		}
		//hipEvent_t start, stop;
		/*std::cout << "Simplex -AFTER CREATION OF Z\n";
		 for (k = 0; k < N_S; k++) {
		 int basen = k * M * nc;
		 for (i = 0; i < M; i++) {
		 for (j = 0; j < nc; j++) {
		 std::cout << N_MAT[(i + j * M) + basen] << "  ";
		 }
		 std::cout << "\n";
		 }
		 std::cout << "\n";
		 } */
//		std::cout << "Before Kernel Called 1\n";
		hipMalloc((void **) &G_MAT, (N_S * M * nc * sizeof(float)));
		//printf("CUDA malloc G_MAT: %s\n", hipGetErrorString(err));
		hipMemcpy(G_MAT, N_MAT, (N_S * M * nc * sizeof(float)),
				hipMemcpyHostToDevice);
		//printf("CUDA malloc N_MAT: %s\n", hipGetErrorString(err));
		//	hipMemcpy(G_Sel, Sel, sizeof(int), hipMemcpyHostToDevice);
		//printf("CUDA malloc G_Sel: %s\n", hipGetErrorString(err));

		//mykernel<<<N_S, threads_per_block>>>(G_MAT, M, nc, G_R, N_S, G_Sel, R_data, R_index);
		mykernel<<<N_S, threads_per_block>>>(G_MAT, M, nc, G_R, N_S, R_data,R_index);
		hipDeviceSynchronize();
		hipMemcpy(R, G_R, N_S * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(N_MAT, G_MAT, (N_S * M * nc * sizeof(float)),hipMemcpyDeviceToHost);
	//for (int k = 0; k < N_S; k
	/*std::cout<< "***********Final SIMPLEX from GPU*************\n Time took:\n";
		 for (int k = 0; k < N_S; k++) {
		 // base=k*M*N;
		 int basen = k * M * nc;
		 for (int i = 0; i < M; i++) {
		 for (int j = 0; j < nc; j++) {
		 std::cout << N_MAT[(i + j * M) + basen] << "\t";
		 }
		 std::cout << "\n";
		 }
		 std::cout << "\n";
		 }		
		*/
		//	std::cout << "Result for Artificial\n";
#pragma omp parallel for
		for (i = 0; i < N_S; i++) {
			int base = i * M * N;
			int basen = i * M * nc;
			for (int j = 0; j < M; j++) {
				for (int k = 0; k < N; k++) {
					if (N_MAT[j + basen] == k + 1) {
						N_MAT[j + M + basen] =	MAT[(M - 1) + (2 + k) * M + base];
					}
				}
			}
		}
		int s;

//std::cout<<"\nResult = "<<R[0]<<"\n";
#pragma omp parallel for
		for (s = 0; s < N_S; s++) {
			if ((roundf(R[s]/10000)*10000) == 0) {
				//std::cout<<"\nInside ** IF*** Result = "<<R[0]<<"\n";
				//int sum = 0;
				int base = s * M * N;
				int basen = s * M * nc;
				for (int i = 0; i < N; i++) {
					float sum = 0;
					for (int j = 0; j < M; j++) {
						if ((j < M - 1)) {
							if (i != (N - 1)) {
								//std::cout<<N_MAT[(j+(i*M))+basen]<<"*"<<N_MAT[(j+(1*M))+basen]<<std::endl;
								sum = sum + (N_MAT[(j + (i * M)) + basen] * N_MAT[(j + (1 * M)) + basen]);
								MAT[(j + (i * M)) + base] = N_MAT[(j + (i * M)) + basen];
							} else if (i == N - 1) {
								sum = sum + (N_MAT[(j + (nc - 1) * M) + basen]
												* N_MAT[(j + (1 * M)) + basen]);
								MAT[(j + (i * M)) + base] = N_MAT[(j + (nc - 1) * M) + basen];
							}
						}
						if (j == (M - 1)) {
							if (i > 1) {
								//std::cout<<sum<<" And "<<MAT[(j+(i*M))+base]<<std::endl;
								MAT[(j + (i * M)) + base] = MAT[(j + (i * M)) + base] + (-1) * sum;
							}
						}
					}
				}
			}
		}
		hipFree (G_MAT);
		//		hipFree(G_R);
		//hipFree(G_Sel);
		//hipDeviceSynchronize();
		//		hipMalloc((void **) &G_R, N_S * sizeof(float));
		hipMalloc((void **) &G_MAT, (N_S * M * N * sizeof(float)));
		// printf("CUDA malloc G_MAT: %s\n", hipGetErrorString(err));
		hipMemcpy(G_MAT, MAT, (N_S * M * N * sizeof(float)),
				hipMemcpyHostToDevice);
		//printf("CUDA malloc N_MAT: %s\n", hipGetErrorString(err));
		//	hipMemcpy(G_Sel, Sel, sizeof(int), hipMemcpyHostToDevice);
		//printf("CUDA malloc G_Sel: %s\n", hipGetErrorString(err));
//std::cout<<"Kernel Called 2\n";

		//mykernel<<<N_S, threads_per_block>>>(G_MAT, M, N, G_R, N_S, G_Sel, R_data, R_index);
		mykernel<<<N_S, threads_per_block>>>(G_MAT, M, N, G_R, N_S, R_data,
				R_index);
		hipDeviceSynchronize();
		hipMemcpy(R, G_R, N_S * sizeof(float), hipMemcpyDeviceToHost);

		/*hipMemcpy(MAT, G_MAT, (N_S * M * N * sizeof(float)),
		 hipMemcpyDeviceToHost);*/
		//std::cout
		// << "***********Final SIMPLEX from GPU*************\n Time took:\n";*/
	}
	//hipFree(G_MAT);
	//hipFree(G_Sel);
	//hipFree(G_R);
	hipFree(R_index);	//Only to synchronise with the cudamemcpy
	//hipFree(R_data);	//Only to synchronise with the cudamemcpy
}

