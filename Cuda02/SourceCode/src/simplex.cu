#include "hip/hip_runtime.h"
#include "simplex.cuh"
#include<omp.h>
#include "iostream"

//LPC without Stream
//Tested Working for Both Helicopter and Five Dimensional system with Reduction
//Both Reduction implemented- One for finding Pivot column and the other for finding Pivot Row
//Implemented Reduction. But without Streams.

// 1st Method:: Most negative value (//Not Working for Large arguments)
__global__ void mykernelOld(float *S_MAT, int S_row, int S_col, float *Result,
		int S_N, float *R_data, int *R_index) {
	//int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int index = blockIdx.x;

	if (index < S_N) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		__shared__ float col1[1024];	//1024 is required to function well for threads
		/*************/
		if (threadIdx.x == 0) {
			c = false;
			rm = 0;
			row = -1;		//pivotRow
			pivotCol = -1;			//printf("Before starting While Loop\n");
		}
		__syncthreads();
		while (!c) {
			__syncthreads();
			int Last_row = S_row - 1;
			//__syncthreads();
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			//__shared__ bool notEntered;
			__shared__ int notEntered;
//not in use	__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
//not in use	minValue = 0;
				newpivotcol = -1;
				//notEntered = true;
				notEntered = 1;
				c = true;
				//printf("Before starting reduction 1\n");
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			int data_size = blockDim.x;
			tid = threadIdx.x;
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) {
				//tid = threadIdx.x - 2;//here my tid should be from 0 to (evenThreadId - 1)
				temp_index = Last_row + tid * S_row + base;	//avoiding re-computation
				R_data[tid + R_base] = S_MAT[temp_index];	//	g_data[i];
				R_index[tid + R_base] = tid;//tid; should be the real index of the data
			} else {
				R_data[tid + R_base] = INT_MAX;	//	g_data[i];
				R_index[tid + R_base] = tid;	//tid;
			}
			__syncthreads();//here will have all values in shared memory from 0 to BLOCK_SIZE

			tid = threadIdx.x;
			for (i = (data_size / 2); i > 0;) {
				if (tid < i) {
					if (R_data[tid + R_base] > R_data[tid + R_base + i]) { //is right-side value small?
						if (R_data[tid + R_base + i] <= -0.000001) {	//only if the value on the right-side is -ive
							R_data[tid + R_base] = R_data[tid + R_base + i];//put the smaller value to left-side
							R_index[tid + R_base] = R_index[tid + R_base + i];

							//notEntered = false;  //race condition avoided
							//notEntered = 0;  //race condition avoided

							int local_notEntered;
							local_notEntered = *(volatile int*) &notEntered;
							atomicCAS(&notEntered, local_notEntered, 0);

						}
					}
				}
				__syncthreads();
				i >>= 1;
				if ((i != 1) && (i % 2) != 0) {	//if s is odd
					i = i + 1;
				}
			}
			// if (notEntered == false && tid == 2) { // tid==0 is always true if minValue is still -1 then what?
			if (threadIdx.x == 0) { // tid==0 is always true if minValue is still -1 then what?
				if (notEntered == false) {
//not in use		minValue = R_data[R_base];
					newpivotcol = R_index[R_base];
					//printf("\nminValue = %f newpivotcol = %d ", minValue,newpivotcol);
				}
			}
			__syncthreads(); //waiting for all threads to have same newpivotcol value
			/*if (threadIdx.x == 0) {
				printf("May be Finished reduction 1\n");
			}
			__syncthreads();*/
			if (newpivotcol == -1) {//All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2;
			} else {
				// ********** Second Reduction Process ******
				//in order to avoid global memory transfer:: Using the same R_data and R_index global memory
				__shared__ float row_min;
				__shared__ int row_num;
				//__shared__ bool notEntered2;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					//notEntered2 = true;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
				//if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
				if (threadIdx.x < Last_row) { //because threadID is from 0
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					if ((S_MAT[temp_index2] > 0) && (S_MAT[temp_index1] > 0)) {
						R_data[k1 + R_base] = S_MAT[temp_index1]
								/ S_MAT[temp_index2];
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //here have all values from 0 to BLOCK_SIZE
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size
				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] >= R_data[indexValue2 + s]) {
							R_data[indexValue2] = R_data[indexValue2 + s];
							R_index[indexValue2] = R_index[indexValue2 + s];

							//notEntered2 = false;
							//notEntered2 = 0;//check using atomic
							int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);

						}
					}
//					if (tid == 0)
//						printf("Data_size = %d ", s);
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						//printf("R_Data = %f R_Index = %d", R_data[R_base], R_index[R_base]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						//if (notEntered2 == true) {
						//return -1;
						//printf("%f ", R_data[R_base]);
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						//}else {
						//return row_num;
						//printf("%f %d ", row_min, row_num);
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************

			//col = S_Sel[index];
			//col = *S_Sel;
			col = pivotCol;
			if (row > -1) {
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					if (tid == remember[tid - 2]) {
						temp_index = (S_row - 1) + (tid * S_row) + base; //avoiding re-computation
						S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}
				}		//Data Parallel section 1 done
				__syncthreads();
				tid = threadIdx.x;
	//			if (threadIdx.x >= 0 && threadIdx.x < S_row) {
				if (threadIdx.x < S_row) { //because threadID is from 0
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();

				unsigned int temp_row_base = row + base;//avoiding re-computation
				S_MAT[temp_row_base + S_row] =
						S_MAT[temp_row_base + col * S_row];
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[row + base] = col - 1;//now temp_row_base is not required
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = S_MAT[temp_index] / S_MAT[row_base + S_row];
				}		//Data Parallel section 3 done
				__syncthreads();
				//printf("Row here = %d",row);
				tid = threadIdx.x;
				//if (threadIdx.x >= 0 && threadIdx.x < S_row) {
				if (threadIdx.x < S_row) {	//because threadID is from 0
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							S_MAT[temp_index] = S_MAT[temp_index]
									- (col1[tid] * S_MAT[row + temp_index1]);
						} else {
							break;
						}
					}
				}	//Data Parallel section 4 done
				__syncthreads();

				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				//tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[((S_row - 1) + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

			} else if (row == -1) {
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col;
					rm++;
				}
				__syncthreads();

				temp_index = (S_row - 1) + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive
				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				// tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[((S_row - 1) + i * S_row) + base] < 0)) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}
		} //end of while
		__syncthreads();
		if (threadIdx.x == 0) {
			//printf("Value = %f ",S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base]);
			Result[index] = S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base];
			//printf("Result = %f  ",Result[index]);
		}
	}
}

//1st Method : Most Negative Value approach (//Works even for Large arguments)
__global__ void mykernelSSS(float *S_MAT, int S_row, int S_col, float *Result,
		int S_N, float *R_data, int *R_index) {
	//int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int index = blockIdx.x;
	if (index < S_N) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		__shared__ float col1[1024];	//pivotColumn
		/*************/
		if (threadIdx.x == 0) {
			c = false;
			rm = 0;
			row = -1;		//pivotRow
			pivotCol = -1;
		}
		__syncthreads();
		while (!c) {
			__syncthreads();
			int Last_row = S_row - 1;
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			__shared__ int notEntered;
			__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
				minValue = 0;
				newpivotcol = -1;
				notEntered = 1;
				c = true;
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			// Since keeping limit only upto (S_col - 1) which is not equal to BLOCK_SIZE creates problem
			// in using syncthreads() inside Reduction for-loop so use all threads(all R_data)
			//int data_size = (S_col - 1) - 2;
			int data_size = blockDim.x;
			tid = threadIdx.x;
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) {
				//tid = threadIdx.x - 2;//here my tid should be from 0 to (evenThreadId - 1)
				//int j = threadIdx.x;//is the actual column/index number less than (S_col - 1)
				//printf("Data_size = %d ", data_size);
				temp_index = Last_row + tid * S_row + base;	//avoiding re-computation
				R_data[tid + R_base] = S_MAT[temp_index];	//	g_data[i];
				R_index[tid + R_base] = tid;//tid; should be the real index of the data
			} else {
				R_data[tid + R_base] = INT_MAX;	//	g_data[i];
				R_index[tid + R_base] = tid;	//tid;
			}
			__syncthreads();//here will have all values in shared memory from 0 to BLOCK_SIZE

			tid = threadIdx.x;
			for (i = (data_size / 2); i > 0;) {
				if (tid < i) {
					//	if ((R_data[tid] >= R_data[tid + s]) && ((R_data[tid + s] < 0) && (R_data[tid] < 0))){
					//(R_data[tid + R_base] < 0) && (R_data[tid + R_base + i] < 0)&&
					if (R_data[tid + R_base] > R_data[tid + R_base + i]) { //is right-side value small?

						//if (R_data[tid + R_base + i] == -0.000000)
							 //R_data[tid + R_base + i] = 0.0;
						//This modification was required for large batch-size(1500) for LP dim 300 and above
						if (R_data[tid + R_base + i] <= -0.000001 ) {	//only if the value on the right-side is -ive
							R_data[tid + R_base] = R_data[tid + R_base + i];//put the smaller value to left-side
							R_index[tid + R_base] = R_index[tid + R_base + i];

							//notEntered = false;  //race condition avoided

							//notEntered = 0;  //race condition avoided
							int local_notEntered;
							local_notEntered = *(volatile int*) &notEntered;
							atomicCAS(&notEntered, local_notEntered, 0);

						}

					}
				}
				/*if (tid == 0)
				 printf("Data_size = %d ", i);*/
				__syncthreads();
				i >>= 1;
				if ((i != 1) && (i % 2) != 0) {	//if s is odd
					i = i + 1;
				}
			}
			// if (notEntered == false && tid == 2) { // tid==0 is always true if minValue is still -1 then what?
			if (threadIdx.x == 0) { // tid==0 is always true if minValue is still -1 then what?
				if (notEntered == false) {
					minValue = R_data[R_base];
					newpivotcol = R_index[R_base];
					//printf("\nminValue = %f newpivotcol = %d ", minValue,newpivotcol);
				}
			}
			__syncthreads(); //waiting for all threads to have same newpivotcol value
			if (newpivotcol == -1) {//All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2;
			} else {
				// ********** Second Reduction Process ******
				//in order to avoid global memory transfer:: Using the same R_data and R_index global memory
				__shared__ float row_min;
				__shared__ int row_num;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
				if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					//This modification was required for large batch-size(5000) for LP dim 300 and above
					if ((S_MAT[temp_index2] >= -0.000001) && (S_MAT[temp_index1] >= -0.000001)) {
						R_data[k1 + R_base] = S_MAT[temp_index1]/ S_MAT[temp_index2];
						//R_data[k1 + R_base] = roundf(((S_MAT[temp_index1]/ S_MAT[temp_index2])/1000000)*1000000);
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //here have all values from 0 to BLOCK_SIZE
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size
				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] >= R_data[indexValue2 + s]) {
							R_data[indexValue2] = R_data[indexValue2 + s];
							R_index[indexValue2] = R_index[indexValue2 + s];
							//notEntered2 = false;

							//notEntered2 = 0;
							int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);
						}
					}
//					if (tid == 0)
//						printf("Data_size = %d ", s);
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						//printf("R_Data = %f R_Index = %d", R_data[R_base], R_index[R_base]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						//if (notEntered2 == true) {
						//return -1;
						//printf("%f ", R_data[R_base]);
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						//}else {
						//return row_num;
						//printf("%f %d ", row_min, row_num);
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************

			//col = S_Sel[index];
			//col = *S_Sel;
			col = pivotCol;
			//printf("Row= %d col = %d\n",row,col);
			if (row > -1) {
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					if (tid == remember[tid - 2]) {
						temp_index = (S_row - 1) + (tid * S_row) + base; //avoiding re-computation
						//if (S_MAT[temp_index] == -0.0)
							//S_MAT[temp_index] = -1 * 0.0; //replacing back to original
						//else
							S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}
				}		//Data Parallel section 1 done
				__syncthreads();
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();

				unsigned int temp_row_base = row + base;//avoiding re-computation
				S_MAT[temp_row_base + S_row] =
						S_MAT[temp_row_base + col * S_row];
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[row + base] = col - 1;//now temp_row_base is not required
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = roundf(((S_MAT[temp_index] / col1[row])/1000000)*1000000);//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = S_MAT[temp_index] / S_MAT[row_base + S_row];
				}		//Data Parallel section 3 done
				__syncthreads();
				//printf("Row here = %d",row);
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							float zeroTemp;
							zeroTemp = col1[tid] * S_MAT[row + temp_index1];
							S_MAT[temp_index] = S_MAT[temp_index] - zeroTemp;
							//S_MAT[temp_index] = roundf(((S_MAT[temp_index] - zeroTemp)/1000000)*1000000);
							//if (S_MAT[temp_index] == -0.0)
								//S_MAT[temp_index]= 0.0;

						} else {
							break;
						}
					}
				}	//Data Parallel section 4 done
				__syncthreads();

				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				//tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[((S_row - 1) + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

			} else if (row == -1) {
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col;
					rm++;
				}
				__syncthreads();

				temp_index = (S_row - 1) + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				//if (S_MAT[temp_index] == -0.0)
					//S_MAT[temp_index] = 0;	//remembering by making positive
				//else
					S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive
				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				// tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[((S_row - 1) + i * S_row) + base] < 0)) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}

		} //end of while
		__syncthreads();

		if (threadIdx.x == 0) {
			//printf("Value = %f ",S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base]);
			Result[index] = S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base];
			//printf("\nResult Inside Kernel: %f \n",Result[index]);
		}
	}
}

//1st Method : Most Negative Value approach
__global__ void mykernel(float *S_MAT, int S_row, int S_col, float *Result,
		int S_N, float *R_data, int *R_index) {
	//int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int index = blockIdx.x;
	if (index < S_N) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		__shared__ float col1[1024];	//pivotColumn
		/*************/
		if (threadIdx.x == 0) {
			c = false;
			rm = 0;
			row = -1;		//pivotRow
			pivotCol = -1;
		}
		__syncthreads();
		while (!c) {
			__syncthreads();
		/*if(threadIdx.x==0){
			for(int ik=0;ik<S_row;ik++){
					for(int jk=0;jk<S_col;jk++){
						printf("%f  ",S_MAT[ik+jk*S_row+index]);
					}
					printf("\n");
				}
			printf("\n\n\n\n");
			printf("Row= %d col = %d\n",row,pivotCol);
			printf("\n\n\n\n");

		}
		__syncthreads(); */
			int Last_row = S_row - 1;
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			__shared__ int notEntered;
			__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
				minValue = 0;
				newpivotcol = -1;
				notEntered = 1;
				c = true;
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			// Since keeping limit only upto (S_col - 1) which is not equal to BLOCK_SIZE creates problem
			// in using syncthreads() inside Reduction for-loop so use all threads(all R_data)
			//int data_size = (S_col - 1) - 2;
			int data_size = blockDim.x;
			tid = threadIdx.x;
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) {
				//tid = threadIdx.x - 2;//here my tid should be from 0 to (evenThreadId - 1)
				//int j = threadIdx.x;//is the actual column/index number less than (S_col - 1)
				//printf("Data_size = %d ", data_size);
				temp_index = Last_row + tid * S_row + base;	//avoiding re-computation
				R_data[tid + R_base] = S_MAT[temp_index];	//	g_data[i];
				R_index[tid + R_base] = tid;//tid; should be the real index of the data
			} else {
				R_data[tid + R_base] = INT_MAX;	//	g_data[i];
				R_index[tid + R_base] = tid;	//tid;
			}
			__syncthreads();//here will have all values in shared memory from 0 to BLOCK_SIZE

			tid = threadIdx.x;
			for (i = (data_size / 2); i > 0;) {
				if (tid < i) {
					//	if ((R_data[tid] >= R_data[tid + s]) && ((R_data[tid + s] < 0) && (R_data[tid] < 0))){
					//(R_data[tid + R_base] < 0) && (R_data[tid + R_base + i] < 0)&&
					if (R_data[tid + R_base] > R_data[tid + R_base + i]) { //is right-side value small?

						//if (R_data[tid + R_base + i] == -0.000000)
							 //R_data[tid + R_base + i] = 0.0;
						if (R_data[tid + R_base + i] <= -0.000001 ) {	//only if the value on the right-side is -ive
							R_data[tid + R_base] = R_data[tid + R_base + i];//put the smaller value to left-side
							R_index[tid + R_base] = R_index[tid + R_base + i];

							//notEntered = false;  //race condition avoided

							//notEntered = 0;  //race condition avoided
							int local_notEntered;
							local_notEntered = *(volatile int*) &notEntered;
							atomicCAS(&notEntered, local_notEntered, 0);

						}

					}
				}
				/*if (tid == 0)
				 printf("Data_size = %d ", i);*/
				__syncthreads();
				i >>= 1;
				if ((i != 1) && (i % 2) != 0) {	//if s is odd
					i = i + 1;
				}
			}
			// if (notEntered == false && tid == 2) { // tid==0 is always true if minValue is still -1 then what?
			if (threadIdx.x == 0) { // tid==0 is always true if minValue is still -1 then what?
				if (notEntered == false) {
					minValue = R_data[R_base];
					newpivotcol = R_index[R_base];
					//printf("\nminValue = %f newpivotcol = %d ", minValue,newpivotcol);
				}
			}
			__syncthreads(); //waiting for all threads to have same newpivotcol value
			//		}
			//		__syncthreads();	//here we have min and newpivotcol
			// ********* First Reduction Ends *************
			//  ******** Second Reduction Begins **********
			/*			if (threadIdx.x == 0) {
			 if (newpivotcol == -1) {
			 //return -2;
			 row = -2;
			 } else {
			 float row_min = INT_MAX;
			 float row_num = -1;
			 //TODO:: this temp_res can be an array of value computed in parallel
			 //TODO:: row_min and row_num can then be computed using reduction
			 for (i = 0; i < S_row - 1; i++) {

			 temp_index = newpivotcol * S_row + i + base; //avoiding re-computation
			 temp_index1 = i + (S_col - 1) * S_row + base; //avoiding re-computation
			 if ((S_MAT[temp_index] > 0)
			 && (S_MAT[temp_index1] > 0)) {
			 float temp_res = S_MAT[temp_index1]
			 / S_MAT[temp_index]; //avoiding re-computation
			 if (temp_res <= row_min) {
			 row_min = temp_res;
			 row_num = i;
			 }
			 }
			 }
			 // *S_Sel = newpivotcol;
			 pivotCol = newpivotcol;
			 //S_Sel[index] = newpivotcol;
			 if (row_min == INT_MAX) {
			 //return -1;
			 row = -1;
			 }
			 if (row_num != -1) {
			 //return row_num;
			 row = row_num;
			 }
			 }
			 } //end of one thread
			 __syncthreads();*/
			if (newpivotcol == -1) {//All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2;
			} else {
				// ********** Second Reduction Process ******
				//in order to avoid global memory transfer:: Using the same R_data and R_index global memory
				__shared__ float row_min;
				__shared__ int row_num;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
				if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					if ((S_MAT[temp_index2] > 0) && (S_MAT[temp_index1] > 0)) {
						R_data[k1 + R_base] = S_MAT[temp_index1]/ S_MAT[temp_index2];
						//R_data[k1 + R_base] = roundf(((S_MAT[temp_index1]/ S_MAT[temp_index2])/1000000)*1000000);
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //here have all values from 0 to BLOCK_SIZE
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size
				/*if (threadIdx.x == 0) {
				 printf("\nR_data \n");
				 for (int x = 0; x < Last_row; x++)
				 printf("%f  ", R_data[x]);
				 printf("\nR_Index \n");
				 for (int x = 0; x < Last_row; x++)
				 printf("%d  ", R_index[x]);
				 printf("Data_size2 = %d ", data_size2);
				 }
				 __syncthreads();*/
				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] >= R_data[indexValue2 + s]) {
							R_data[indexValue2] = R_data[indexValue2 + s];
							R_index[indexValue2] = R_index[indexValue2 + s];
							//notEntered2 = false;

							//notEntered2 = 0;
							int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);
						}
					}
//					if (tid == 0)
//						printf("Data_size = %d ", s);
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						//printf("R_Data = %f R_Index = %d", R_data[R_base], R_index[R_base]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						//if (notEntered2 == true) {
						//return -1;
						//printf("%f ", R_data[R_base]);
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						//}else {
						//return row_num;
						//printf("%f %d ", row_min, row_num);
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************

			//col = S_Sel[index];
			//col = *S_Sel;
			col = pivotCol;
			//printf("Row= %d col = %d\n",row,col);
			if (row > -1) {
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					if (tid == remember[tid - 2]) {
						temp_index = (S_row - 1) + (tid * S_row) + base; //avoiding re-computation
						//if (S_MAT[temp_index] == -0.0)
							//S_MAT[temp_index] = -1 * 0.0; //replacing back to original
						//else
							S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}
				}		//Data Parallel section 1 done
				__syncthreads();
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();

				unsigned int temp_row_base = row + base;//avoiding re-computation
				S_MAT[temp_row_base + S_row] =
						S_MAT[temp_row_base + col * S_row];
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[row + base] = col - 1;//now temp_row_base is not required
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = roundf(((S_MAT[temp_index] / col1[row])/1000000)*1000000);//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = S_MAT[temp_index] / S_MAT[row_base + S_row];
				}		//Data Parallel section 3 done
				__syncthreads();
				//printf("Row here = %d",row);
				tid = threadIdx.x;
				if (threadIdx.x >= 0 && threadIdx.x < S_row) {
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							float zeroTemp;
							zeroTemp = col1[tid] * S_MAT[row + temp_index1];
							S_MAT[temp_index] = S_MAT[temp_index] - zeroTemp;
							//S_MAT[temp_index] = roundf(((S_MAT[temp_index] - zeroTemp)/1000000)*1000000);
							//if (S_MAT[temp_index] == -0.0)
								//S_MAT[temp_index]= 0.0;

						} else {
							break;
						}
					}
				}	//Data Parallel section 4 done
				__syncthreads();

				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				//tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[((S_row - 1) + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

			} else if (row == -1) {
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col;
					rm++;
				}
				__syncthreads();

				temp_index = (S_row - 1) + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				//if (S_MAT[temp_index] == -0.0)
					//S_MAT[temp_index] = 0;	//remembering by making positive
				//else
					S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive
				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				// tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[((S_row - 1) + i * S_row) + base] < 0)) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}

		} //end of while
		__syncthreads();

		if (threadIdx.x == 0) {
	/*
				if(threadIdx.x==0){
			for(int ik=0;ik<S_row;ik++){
					for(int jk=0;jk<S_col;jk++){
						printf("%f  ",S_MAT[ik+jk*S_row+index]);
					}
					printf("\n");
				}
			printf("\n\n\n\n");
			printf("Row= %d col = %d\n",row,pivotCol);
			printf("\n\n\n\n");

		}
		__syncthreads();
			//printf("Value = %f ",S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base]);

*/			Result[index] = S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base];
			//printf("\nResult Inside Kernel: %f \n",Result[index]);
		}
	}
}


__host__ Simplex::Simplex(unsigned int N_S) {
	number_of_LPs = N_S;
	//i = 0;
	//a = 0.0;
	M = 0;
	N = 0;
	//NB = 0;
	c = 0;
	No_c = 0;
	//f = j = 0;
	//R = (float*) malloc(N_S * sizeof(float));
	R = (float*) calloc(N_S, sizeof(float));

}

//get status of particular simplex
__host__ int Simplex::getStatus(int n) {
	int s;
	for (int i = 0; i < C.size1(); i++) {
		if (i == (n - 1)) {
			if (R[i] == -1) {
				s = 6;	// 6 = Simplex Is Unbounded
			} else if (R[i] > 0) {
				s = 2;	// 2= Simplex has feasible and Optimal solution
			}
		}
	}
	return s;

}	//get status of particular simplex

//get the No of simplex the object is ruuning on GPU
__host__ int Simplex::getNo_OF_Simplx() {
	return C.size1();
}	//get the No of simplex the object is ruuning on GPU

//get the result of all simplex
__host__ std::vector<float> Simplex::getResultAll() {

	std::vector<float> Res(C.size1());
	for (int i = 0; i < C.size1(); i++) {
		Res[i] = R[i];
	}
	return Res;
}

//get the result of all simplex

__host__ float Simplex::getResult(int n) {
	// get result of particular simplex
	float r;
	for (int i = 0; i < C.size1(); i++) {
		if (i == (n - 1)) {
			r = R[i];
		}
	}
	return r;
}	// get result of particular simplex

__host__ std::vector<int> Simplex::getStatusAll() {

	std::vector<int> Status(C.size1());
	for (int i = 0; i < C.size1(); i++) {
		if (R[i] == -1)
			Status[i] = 6;
		else
			Status[i] = 2;
	}
	return Status;
}	//get the status of all simplex

__host__ void Simplex::setConstratint(math::matrix<double> A,
		std::vector<double> B) {
	int N_S = number_of_LPs;
	orig_CoefficientMatrix = A;
	BoundValue = B;
//	std::cout<<"Before setConstraints called\n";
//	A = math::matrix<float>(A1);
//	B = std::vector<float>(B1);
	int No_O = A.size2();
	//std::cout << "No of Variable is " << A.size2() << " And no of constraints "	<< A.size1() << std::endl;
	int No_C = A.size1();
	M = No_C + 1;
	N = No_O + 3 + No_C;
	c = 1 + No_O;
	//NB = c;
	//f = 0;

	/*Sel = (int *) malloc(N_S * sizeof(int));
	 R = (float*) malloc(N_S * sizeof(float));*/
	//MAT = (float *) calloc(N_S * M * N, sizeof(float));

	MAT_COPY = (float *) calloc(N_S * M * N, sizeof(float));

	/*hipMalloc((void **) &G_MAT, (N_S * M * N * sizeof(float)));
	 hipMalloc((void **) &G_R, N_S * sizeof(float));
	 hipMalloc((void **) &G_Sel, N_S * sizeof(int));*/
int s;
#pragma omp parallel for
	for (s = 0; s < N_S; s++) {
		unsigned int some = M * N * s;
		for (int i = 0; i < M - 1; i++) {
			for (int j = 0; j < N; j++) {
				if (j == 0) {
					//MAT[(int) ((i + j * M) + (M * N * s))] = c + i;
					//MAT[(int) ((i + j * M) + some)] = c + i;
					MAT_COPY[(int) ((i + j * M) + some)] = c + i;
				} else if (j > 1) {
					if (j < (No_O + 2)) {	//Coefficient of A
						//MAT[(int) ((i + j * M) + (M * N * s))] = (float) A(i, j - 2);
						//MAT[(int) ((i + j * M) + some)] = (float) A(i, j - 2);
						MAT_COPY[(int) ((i + j * M) + some)] = (float) A(i, j - 2);
					} else if (j == N - 1) {//std::cout<<"Enter RHS of coefficient "<< i+1 <<"\n";
						//MAT[(int) ((i + j * M) + (M * N * s))] = (float) B[i];
						//MAT[(int) ((i + j * M) + some)] = (float) B[i];
						MAT_COPY[(int) ((i + j * M) + some)] = (float) B[i];
					} else if (j < N - 1) {
						//MAT[(int) ((i + (No_O + 2 + i) * M) + (M * N * s))] = 1;
						//MAT[(int) ((i + (No_O + 2 + i) * M) + some)] = 1;
						MAT_COPY[(int) ((i + (No_O + 2 + i) * M) + some)] = 1;
					}
				}
			}
		}
	}
//	std::cout<<"setting constraints of simplex Done\n";
}	//setting constraints of simplex

__host__ void Simplex::ComputeLP(math::matrix<float> &C1) {

	hipError_t err;

	unsigned int threads_per_block;	//Maximum threads depends on CC 1.x =512 2.x and > = 1024

	unsigned int number_of_blocks;//depends on our requirements (better to be much more than the number of SMs)

	int device;
	hipDeviceProp_t props;
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device);

	int No_C = orig_CoefficientMatrix.size1();
	C = math::matrix<float>(C1);

	int N_S = C.size1();

	int No_O = C.size2();
	M = No_C + 1, N = No_O + 3 + No_C;
	int N_C = No_C;
	c = 1 + No_O;
	//float sum = 0;
	int s;
#pragma omp parallel for
	for (s = 0; s < N_S; s++) {
		unsigned int some = M * N * s;
		for (int i = M - 1; i < M; i++) {
			for (int j = 2; j < N; j++) {
				if (j < 2 + No_O) {
					//MAT[(int) ((i + j * M) + (M * N * s))] = -C(s, j - 2);
					//MAT[(int) ((i + j * M) + some)] = -C(s, j - 2);
					MAT_COPY[(int) ((i + j * M) + some)] = -C(s, j - 2);
				}
			}
		}
	}
	std::vector<int> rem;
	for (int i = 0; i < N_C; i++) {
		//std::cout << B[i] << "\n";
		if (BoundValue[i] < 0) {
			rem.push_back(i);
			//count++;
			//std::cout<<B[i]<<"\n";
		}
	}

	//std::cout<<"C= "<< rem.size()<<"\n";
	int nc = N + rem.size();
	threads_per_block = 32 * (nc / 32) + 32; //if count equal 0 than nc=N so works for for Model
	if (threads_per_block > props.maxThreadsPerBlock) //Assuming maximum threads supported by CC is 1024
		threads_per_block = props.maxThreadsPerBlock;

	int *R_index;	//reduction data
	float *R_data;	//reduction index
	err = hipMalloc((void **) &R_data,
			C1.size1() * threads_per_block * sizeof(float));//C1.size1() * 96 being the maximum threads
	err = hipMalloc((void **) &R_index,
			C1.size1() * threads_per_block * sizeof(int));//C1.size1() being the number of LPs

	err = hipMalloc((void **) &G_R, N_S * sizeof(float));//Doing it here for the First Time

	//printf("CUDA malloc R_index: %s\n", hipGetErrorString(err));
	//std::cout << "Number of threads per block = " << threads_per_block << "\n";

	if (rem.size() > 0) {	//Helicopter model has no negative bound so count=0
	;
	} else {
		//hipEvent_t start, stop;

		err = hipMalloc((void **) &G_MAT, (N_S * M * N * sizeof(float)));
		//printf("CUDA malloc G_MAT : %s\n", hipGetErrorString(err));
		//err = hipMemcpy(G_MAT, MAT, (N_S * M * N * sizeof(float)), hipMemcpyHostToDevice);
		err = hipMemcpy(G_MAT, MAT_COPY, (N_S * M * N * sizeof(float)), hipMemcpyHostToDevice);

	//	printf("CUDA memcpy G_MAT : %s\n", hipGetErrorString(err));
//
	//	std::cout << "Size copied(bytes) = "<<(long)(N_S * M * N * sizeof(float));
		//err = hipMemcpy(G_Sel, Sel, sizeof(int), hipMemcpyHostToDevice);
		//printf("CUDA memcpy G_Sel: %s\n", hipGetErrorString(err));
		/*std::cout << "Simplex -AFTER CREATION OF Z\n";

		 for (i = 0; i < M; i++) {
		 for (j = 0; j < N; j++) {
		 std::cout << MAT[(i + j * M)] << "(" << (i + j * M) << ")\t";
		 }
		 std::cout << "\n";
		 }*/
	//	std::cout << "Before Kernel Call\n";
		//mykernel<<<N_S, threads_per_block>>>(G_MAT, M, N, G_R, N_S, G_Sel, R_data, R_index);
		mykernel<<<N_S, threads_per_block>>>(G_MAT, M, N, G_R, N_S, R_data, R_index);
	//	std::cout << "After Kernel Called\n";
		//	hipDeviceSynchronize();		//removed as hopping that hipFree will handle it
		err = hipMemcpy(R, G_R, N_S * sizeof(float), hipMemcpyDeviceToHost);
	//	printf("CUDA memcpy G_R: %s\n", hipGetErrorString(err));
		//hipMemcpy(MAT, G_MAT, (N_S * M * N * sizeof(float)), hipMemcpyDeviceToHost);
		//	std::cout << "N_S = " << N_S << std::endl;
	}
//	std::cout << "before hipFree \n";
//	hipFree(G_MAT);
//	hipFree(G_Sel);
	hipFree(R_index);	//Only to synchronise with the cudamemcpy

//	std::cout << "After hipFree \n";
}

//  Computes the entire list of LPs by diving into different blocks :: this interface not use at present
std::vector<float> Simplex::bulkSolver(math::matrix<float> &List_of_ObjValue) {
	unsigned int tot_lp = List_of_ObjValue.size1();
	std::cout << "Total LPs " << tot_lp << std::endl;
	int lp_block_size = 1000;//input how many LPs you want to solve at a time ??????
	unsigned int number_of_blocks;
	if (tot_lp % lp_block_size == 0)
		number_of_blocks = tot_lp / lp_block_size;
	else
		number_of_blocks = (tot_lp / lp_block_size) + 1;
	std::cout << "Total Blocks " << number_of_blocks << std::endl;

	std::list<block_lp> bulk_lps(number_of_blocks);	//list of sub-division of LPs
	struct block_lp myLPList;
	myLPList.block_obj_coeff.resize(lp_block_size, List_of_ObjValue.size2());
	math::matrix<float> block_obj_coeff(lp_block_size,
			List_of_ObjValue.size2());
	unsigned int index = 0;
	for (unsigned int lp_number = 0; lp_number < tot_lp; lp_number++) {
		for (unsigned int i = 0; i < List_of_ObjValue.size2(); i++) {
			myLPList.block_obj_coeff(index, i) = List_of_ObjValue(lp_number, i);
		}
		index++;
		if (index == lp_block_size) {
			index = 0;
			bulk_lps.push_back(myLPList);
		}
	}	//end of all LPs
	std::list<block_lp_result> bulk_result(number_of_blocks);
	struct block_lp_result eachBlock;
	eachBlock.results.resize(lp_block_size);	//last block will be less

	for (std::list<block_lp>::iterator it = bulk_lps.begin();
			it != bulk_lps.end(); it++) {
		ComputeLP((*it).block_obj_coeff);
		eachBlock.results = this->getResultAll();
		bulk_result.push_back(eachBlock);
	}
	std::vector<float> res(tot_lp);
	unsigned int index_res = 0;
	for (std::list<block_lp_result>::iterator it = bulk_result.begin();
			it != bulk_result.end(); it++) {
		unsigned int block_result_size = (*it).results.size();
		for (unsigned int i = 0; i < block_result_size; i++) {
			res[index_res] = (*it).results[i];
			index_res++;
		}
	}
	std::cout << "Result size = " << res.size() << std::endl;
//R = res;
	return res;
}

