#include "hip/hip_runtime.h"
#include "simplex.cuh"
#include<omp.h>
#include "iostream"

//Both Reduction implemented- One for finding Pivot column and the other for finding Pivot Row
//Implemented Reduction. But without Streams.


// 2nd Method:: Random negative value
__global__ void mykernel(float *S_MAT, int S_row, int S_col, float *Result,
		int S_N, float *R_data, int *R_index) {
	//int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int index = blockIdx.x;
	if (index < S_N) {
		int tid;
		int i; // used for for index
		unsigned int temp_index;
		unsigned int temp_index1;
		int base = index * S_row * S_col;
		int R_base = index * blockDim.x;  // blockDim.x = 96
		__shared__ bool c;
		__shared__ int rm;
		__shared__ int row;	//pivotRow
		__shared__ int pivotCol;//pivotCol this can remove global variable S_Sel

		int col = 1;
		__shared__ int remember[1024]; //Found a column which is negative but theta/Min has no positive value
		__shared__ float col1[1024];	//pivotColumn
		/*************/
		if (threadIdx.x == 0) {
			c = false;
			rm = 0;
			row = -1;		//pivotRow
			pivotCol = -1;
		}
		__syncthreads();
		while (!c) {
			__syncthreads();
			int Last_row = S_row - 1;
			//   ***************** Get_Pivot function begins  *****************
			// ******** First Reduction Begins **********
			//using reduction to compute min and newpivotcol
			__shared__ float minValue;
			__shared__ int newpivotcol;
			if (threadIdx.x == 0) {
				minValue = 0;
				newpivotcol = -1;
				c = true;
			}
			__syncthreads();	//making sure newpivotcol is initialised to -1
			//for (int j = 2; j < S_col - 1; j++) {//only last row but all column
			if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)) {
				int j = threadIdx.x;
				unsigned int temp_index1 = Last_row + j * S_row + base; //avoiding re-computation
				if (S_MAT[temp_index1] < minValue) {
					//minValue = S_MAT[temp_index1];

					//newpivotcol = j; //"Any(Random) negative coefficient rule"

					int local_NewPivotCol;
					local_NewPivotCol = *(volatile int*) &newpivotcol;
					atomicCAS(&newpivotcol, local_NewPivotCol, j);

					/*
					 http://stackoverflow.com/questions/27616417/cuda-is-there-any-way-to-prevent-other-threads-from-changing-a-shared-or-global
					 if (atomicCAS(&newpivotcol, local_NewPivotCol, j)==local_NewPivotCol){
					 //this thread won the write
					 printf("Thread ID = %d ",threadIdx.x);
					 }*/
					//break;
				}
			}
			__syncthreads(); //here we have min and newpivotcol

			//  ******** Second Reduction Begins **********

			if (newpivotcol == -1) { //All Threads will follow the Same path so no issue with divergence
				//return -2;
				row = -2;
			} else {
				// ********** Second Reduction Process ******
				//in order to avoid global memory transfer:: Using the same R_data and R_index global memory
				__shared__ float row_min;
				__shared__ int row_num;
				__shared__ int notEntered2;
				if (threadIdx.x == 0) {
					row_min = INT_MAX;
					row_num = -1;
					notEntered2 = 1;
				}
				__syncthreads();
				// Since keeping limit only upto Last_row which is not equal to block_size creates problem
				// in using syncthreads() inside Reduction for-loop so use all threads(all R_data
				int k1;
			//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {
				if (threadIdx.x < Last_row) { //because threadID is from 0
					k1 = threadIdx.x;	//here k1 =0 to Last_row only
					//for (int k1 = 0; k1 < Last_row; k1++) {	//Last_row = (S_row - 1)
					int temp_index2 = newpivotcol * S_row + k1 + base;
					temp_index1 = k1 + (S_col - 1) * S_row + base; //avoiding re-computation
					if ((S_MAT[temp_index2] > 0) && (S_MAT[temp_index1] > 0)) {
						R_data[k1 + R_base] = S_MAT[temp_index1]
								/ S_MAT[temp_index2];
						R_index[k1 + R_base] = k1;
					} else {
						R_data[k1 + R_base] = INT_MAX; //to make the array size equal
						R_index[k1 + R_base] = k1; //to make the array size equal
					}
				} else { //remaining threads above Last_row(including) upto Block_Size
					k1 = threadIdx.x;
					R_data[k1 + R_base] = INT_MAX; //to make the array size equal
					R_index[k1 + R_base] = k1; //to make the array size equal
				}
				__syncthreads(); //here have all values from 0 to BLOCK_SIZE
				//Now find the minValue and its index from R_data and R_index using Reduction
				//int data_size = Last_row;
				int data_size2 = blockDim.x; //Now it is Block_Size

				// ***** Second Reduction on R_data and R_index ****
				//	if (threadIdx.x >= 0 && threadIdx.x < Last_row) {	//Now for all threads
				tid = threadIdx.x;
				for (int s = (data_size2 / 2); s > 0;) {
					if (tid < s) {
						int indexValue2 = tid + R_base;
						if (R_data[indexValue2] >= R_data[indexValue2 + s]) {
							R_data[indexValue2] = R_data[indexValue2 + s];
							R_index[indexValue2] = R_index[indexValue2 + s];
							//notEntered2 = false;

							//notEntered2 = 0;

							int local_notEntered2;
							local_notEntered2 = *(volatile int*) &notEntered2;
							atomicCAS(&notEntered2, local_notEntered2, 0);

						}
					}
//					if (tid == 0)
//						printf("Data_size = %d ", s);
					__syncthreads();	//This creates unpredictable behaviour
					s >>= 1;
					if ((s != 1) && (s % 2) != 0) {	//if s is odd
						s = s + 1;
					}
				}
				//if (notEntered2 == false && tid == 0) {
				if (tid == 0) {
					if (notEntered2 == false) {
						row_min = R_data[R_base];
						row_num = R_index[R_base];
						//printf("R_Data = %f R_Index = %d", R_data[R_base], R_index[R_base]);
					}
				}
				__syncthreads(); // Looks like this can be skipped
				//	}
				//	__syncthreads();	//here we have Row_min and newpivotRow
				// ********** Second Reduction on R_data and R_index ******
				if (threadIdx.x == 0) {
					pivotCol = newpivotcol;
					if (row_min == INT_MAX) {
						//if (notEntered2 == true) {
						//return -1;
						//printf("%f ", R_data[R_base]);
						row = -1;
					}
					if ((row_min != INT_MAX) && (row_num != -1)) {
						//}else {
						//return row_num;
						//printf("%f %d ", row_min, row_num);
						row = row_num;
					}
				}
				__syncthreads(); // Looks like this can be skipped
			} //end of else of newpivotcol == -1
			__syncthreads(); // Looks like this can be skipped but here we have row synchronized
			//  ******** Second Reduction Ends **********
			//   ***************** Get_Pivot function ends  *****************

			//col = S_Sel[index];
			//col = *S_Sel;
			col = pivotCol;
			if (row > -1) {
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int i1 = 2; i1 < S_col; i1++) {		//Data Parallel section 1
					if (tid == remember[tid - 2]) {
						temp_index = (S_row - 1) + (tid * S_row) + base; //avoiding re-computation
						S_MAT[temp_index] = -1 * S_MAT[temp_index]; //replacing back to original
					}
				}		//Data Parallel section 1 done
				__syncthreads();
				tid = threadIdx.x;
				//if (threadIdx.x >= 0 && threadIdx.x < S_row) {
				if (threadIdx.x < S_row) {//because threadID is from 0
					//for (int i = 0; i < S_row; i++) {	//Data Parallel section 2
					col1[tid] = S_MAT[(tid + col * S_row) + base];//keeping the old pivotcol coeff
				}	//Data Parallel section 2 done
				__syncthreads();

				unsigned int temp_row_base = row + base;//avoiding re-computation
				S_MAT[temp_row_base + S_row] =
						S_MAT[temp_row_base + col * S_row];
				//S_MAT[temp_row_base] = col - 1;
				S_MAT[row + base] = col - 1;//now temp_row_base is not required
				tid = threadIdx.x;
				if (threadIdx.x >= 2 && threadIdx.x < S_col) {
					//for (int j = 2; j < S_col; j++){		//Data Parallel section 3
					unsigned int row_base = row + base;	//avoiding re-computation
					temp_index = row_base + (tid * S_row);//avoiding re-computation
					S_MAT[temp_index] = S_MAT[temp_index] / col1[row];//S_MAT[row_base + S_row];
					//S_MAT[temp_index] = S_MAT[temp_index] / S_MAT[row_base + S_row];
				}		//Data Parallel section 3 done
				__syncthreads();
				//printf("Row here = %d",row);
				tid = threadIdx.x;
				//if (threadIdx.x >= 0 && threadIdx.x < S_row) {
				if (threadIdx.x < S_row) {//because threadID is from 0
					//for (int i = 0; i < S_row; i++) {	//Data parallel section 4
					for (i = 2; i < S_col; i++) {
						if (tid != row) {
							temp_index1 = i * S_row + base;
							temp_index = tid + temp_index1;
							S_MAT[temp_index] = S_MAT[temp_index]
									- (col1[tid] * S_MAT[row + temp_index1]);
						} else {
							break;
						}
					}
				}	//Data Parallel section 4 done
				__syncthreads();

				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				//tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {
						if (S_MAT[((S_row - 1) + i * S_row) + base] < 0) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();

			} else if (row == -1) {
				if (threadIdx.x == 0) {
					c = true;
					remember[rm] = col;
					rm++;
				}
				__syncthreads();

				temp_index = (S_row - 1) + (col * S_row) + base; //if col==-1 than problem for base==0 i.e. temp_index==-1
				S_MAT[temp_index] = -1 * S_MAT[temp_index];	//remembering by making positive
				//if (threadIdx.x >= 2 && threadIdx.x < (S_col - 1)){
				// tid = threadIdx.x;
				if (threadIdx.x == 0) {
					for (i = 2; i < (S_col - 1); i++) {		//Data parallel 5
						if ((S_MAT[((S_row - 1) + i * S_row) + base] < 0)) {
							c = false; // check needed for race condition here.
							break;
						}
					}
				}
				__syncthreads();
			}
		} //end of while
		__syncthreads();
		if (threadIdx.x == 0) {
			//printf("Value = %f ",S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base]);
			Result[index] = S_MAT[(S_row - 1 + (S_col - 1) * S_row) + base];
		}
	}
}



__host__ Simplex::Simplex(unsigned int N_S) {
	number_of_LPs = N_S;
	//i = 0;
	//a = 0.0;
	M = 0;
	N = 0;
	//NB = 0;
	c = 0;
	No_c = 0;
	//f = j = 0;
//	Sel = (int *) malloc(N_S * sizeof(int));
	R = (float*) malloc(N_S * sizeof(float));
	/*
	 MAT = (float *) malloc(N_S * M * N * sizeof(float));
	 N_MAT = (float *) malloc(N_S * M * N * sizeof(float));
	 hipMalloc((void **) &G_MAT, (N_S * M * (N + 1) * sizeof(float)));*/
	/*	hipError_t err;
	 err = hipMalloc((void **) &G_R, N_S * sizeof(float));
	 */

	//printf("CUDA malloc G_R: %s\n", hipGetErrorString(err));
//	hipMalloc((void **) &G_Sel, N_S * sizeof(int));
	//printf("CUDA malloc G_Sel: %s\n", hipGetErrorString(err));
}

//get status of particular simplex
__host__ int Simplex::getStatus(int n) {
	int s;
	for (int i = 0; i < C.size1(); i++) {
		if (i == (n - 1)) {
			if (R[i] == -1) {
				s = 6;	// 6 = Simplex Is Unbounded
			} else if (R[i] > 0) {
				s = 2;	// 2= Simplex has feasible and Optimal solution
			}
		}
	}
	return s;

}	//get status of particular simplex

//get the No of simplex the object is ruuning on GPU
__host__ int Simplex::getNo_OF_Simplx() {
	return C.size1();
}	//get the No of simplex the object is ruuning on GPU

//get the result of all simplex
__host__ std::vector<float> Simplex::getResultAll() {

	std::vector<float> Res(C.size1());
	for (int i = 0; i < C.size1(); i++) {
		Res[i] = R[i];
	}
	return Res;
}

//get the result of all simplex

__host__ float Simplex::getResult(int n) {
	// get result of particular simplex
	float r;
	for (int i = 0; i < C.size1(); i++) {
		if (i == (n - 1)) {
			r = R[i];
		}
	}
	return r;
}	// get result of particular simplex

__host__ std::vector<int> Simplex::getStatusAll() {

	std::vector<int> Status(C.size1());
	for (int i = 0; i < C.size1(); i++) {
		if (R[i] == -1)
			Status[i] = 6;
		else
			Status[i] = 2;
	}
	return Status;
}	//get the status of all simplex

__host__ void Simplex::setConstratint(math::matrix<double> A,std::vector<double> B) {
	int N_S = number_of_LPs;
	orig_CoefficientMatrix = A;
	BoundValue = B;
//	std::cout<<"Before setConstraints called\n";
//	A = math::matrix<float>(A1);
//	B = std::vector<float>(B1);
	int No_O = A.size2();
	//std::cout << "No of Variable is " << A.size2() << " And no of constraints "	<< A.size1() << std::endl;
	int No_C = A.size1();
	M = No_C + 1;
	N = No_O + 3 + No_C;
	c = 1 + No_O;
	//NB = c;
	//f = 0;

	/*Sel = (int *) malloc(N_S * sizeof(int));
	 R = (float*) malloc(N_S * sizeof(float));*/
	MAT = (float *) calloc(N_S * M * N, sizeof(float));
	/*hipMalloc((void **) &G_MAT, (N_S * M * N * sizeof(float)));
	 hipMalloc((void **) &G_R, N_S * sizeof(float));
	 hipMalloc((void **) &G_Sel, N_S * sizeof(int));*/
	#pragma omp parallel for
	for (int s = 0; s < N_S; s++) {
		for (int i = 0; i < M-1; i++) {
			for (int j = 0; j < N; j++) {
				if (j == 0) {
					MAT[(int) ((i + j * M) + (M * N * s))] = c+i;
				} else if (j > 1) {
					if (j < (No_O + 2)) {//Coefficient of A
						MAT[(int) ((i + j * M) + (M * N * s))] = (float) A(i, j - 2);
					} else if (j == N - 1) {//std::cout<<"Enter RHS of coefficient "<< i+1 <<"\n";
						MAT[(int) ((i + j * M) + (M * N * s))] = (float) B[i];
					} else if (j < N - 1) {
						MAT[(int) ((i + (No_O+2+i) * M) + (M * N * s))] = 1;
					}
				}
			}
		}
	}
	//std::cout<<"setting constraints of simplex Done\n";
}	//setting constraints of simplex

__host__ void Simplex::ComputeLP(math::matrix<float> &C1) {

	hipError_t err;
	unsigned int threads_per_block;	//Maximum threads depends on CC 1.x =512 2.x and > = 1024

	unsigned int number_of_blocks;//depends on our requirements (better to be much more than the number of SMs)

	int device;
	hipDeviceProp_t props;
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device);

	int No_C = orig_CoefficientMatrix.size1();
	C = math::matrix<float>(C1);

	int N_S = C.size1();

	int No_O = C.size2();
	M = No_C + 1, N = No_O + 3 + No_C;
	int N_C = No_C;
	c = 1 + No_O;
	//float sum = 0;
	#pragma omp parallel for
	for (int s = 0; s < N_S; s++) {
		for (int i = M - 1; i < M; i++) {
			for (int j = 2; j < N; j++) {
				if (j < 2 + No_O) {
					MAT[(int) ((i + j * M) + (M * N * s))] = -C(s, j - 2);
				}
			}
		}
	}
	std::vector <int> rem;
	for (int i = 0; i < N_C; i++) {
		if (BoundValue[i] < 0) {
			rem.push_back(i);
		}
	}

	//std::cout<<"C= "<< rem.size()<<"\n";
	int nc = N + rem.size();
	threads_per_block = 32 * (nc / 32) + 32; //if count equal 0 than nc=N so works for for Model
	if (threads_per_block > props.maxThreadsPerBlock) //Assuming maximum threads supported by CC is 1024
		threads_per_block = props.maxThreadsPerBlock;

	int *R_index;	//reduction data
	float *R_data;	//reduction index
	err = hipMalloc((void **) &R_data,
			C1.size1() * threads_per_block * sizeof(float));//C1.size1() * 96 being the maximum threads
	err = hipMalloc((void **) &R_index,
			C1.size1() * threads_per_block * sizeof(int));//C1.size1() being the number of LPs
	err = hipMalloc((void **) &G_R, N_S * sizeof(float));//Doing it here for the First Time

	//printf("CUDA malloc R_index: %s\n", hipGetErrorString(err));
	//std::cout << "Number of threads per block = " << threads_per_block << "\n";

	if (rem.size() > 0) {
		;
	}	else {
		//hipEvent_t start, stop;

		err = hipMalloc((void **) &G_MAT, (N_S * M * N * sizeof(float)));
		//printf("CUDA malloc G_MAT : %s\n", hipGetErrorString(err));
		err = hipMemcpy(G_MAT, MAT, (N_S * M * N * sizeof(float)),
				hipMemcpyHostToDevice);
		//printf("CUDA memcpy G_MAT : %s\n", hipGetErrorString(err));

	//std::cout << "Size copied(bytes) = "<<(long)(N_S * M * N * sizeof(float));

		//err = hipMemcpy(G_Sel, Sel, sizeof(int), hipMemcpyHostToDevice);
		//printf("CUDA memcpy G_Sel: %s\n", hipGetErrorString(err));
		/*std::cout << "Simplex -AFTER CREATION OF Z\n";

		 for (i = 0; i < M; i++) {
		 for (j = 0; j < N; j++) {
		 std::cout << MAT[(i + j * M)] << "(" << (i + j * M) << ")\t";
		 }
		 std::cout << "\n";
		 }*/
	//	std::cout << "Before Kernel Call\n";
		//mykernel<<<N_S, threads_per_block>>>(G_MAT, M, N, G_R, N_S, G_Sel, R_data, R_index);
		mykernel<<<N_S, threads_per_block>>>(G_MAT, M, N, G_R, N_S, R_data, R_index);
	//	std::cout << "After Kernel Called\n";
		//	hipDeviceSynchronize();		//removed as hopping that hipFree will handle it
		err = hipMemcpy(R, G_R, N_S * sizeof(float), hipMemcpyDeviceToHost);
		//printf("CUDA memcpy G_R: %s\n", hipGetErrorString(err));
		//hipMemcpy(MAT, G_MAT, (N_S * M * N * sizeof(float)), hipMemcpyDeviceToHost);
		//	std::cout << "N_S = " << N_S << std::endl;
	}
//	std::cout << "before hipFree \n";
//	hipFree(G_MAT);
//	hipFree(G_Sel);
	hipFree(R_index);	//Only to synchronise with the cudamemcpy
//	hipDeviceReset();
//	std::cout << "After hipFree \n";
}

//  Computes the entire list of LPs by diving into different blocks :: this interface not use at present
std::vector<float> Simplex::bulkSolver(math::matrix<float> &List_of_ObjValue) {
	unsigned int tot_lp = List_of_ObjValue.size1();
	std::cout << "Total LPs " << tot_lp << std::endl;
	int lp_block_size = 1000;//input how many LPs you want to solve at a time ??????
	unsigned int number_of_blocks;
	if (tot_lp % lp_block_size == 0)
		number_of_blocks = tot_lp / lp_block_size;
	else
		number_of_blocks = (tot_lp / lp_block_size) + 1;
	std::cout << "Total Blocks " << number_of_blocks << std::endl;

	std::list<block_lp> bulk_lps(number_of_blocks);	//list of sub-division of LPs
	struct block_lp myLPList;
	myLPList.block_obj_coeff.resize(lp_block_size, List_of_ObjValue.size2());
	math::matrix<float> block_obj_coeff(lp_block_size,
			List_of_ObjValue.size2());
	unsigned int index = 0;
	for (unsigned int lp_number = 0; lp_number < tot_lp; lp_number++) {
		for (unsigned int i = 0; i < List_of_ObjValue.size2(); i++) {
			myLPList.block_obj_coeff(index, i) = List_of_ObjValue(lp_number, i);
		}
		index++;
		if (index == lp_block_size) {
			index = 0;
			bulk_lps.push_back(myLPList);
		}
	}	//end of all LPs
	std::list<block_lp_result> bulk_result(number_of_blocks);
	struct block_lp_result eachBlock;
	eachBlock.results.resize(lp_block_size);	//last block will be less

	for (std::list<block_lp>::iterator it = bulk_lps.begin();
			it != bulk_lps.end(); it++) {
		ComputeLP((*it).block_obj_coeff);
		eachBlock.results = this->getResultAll();
		bulk_result.push_back(eachBlock);
	}
	std::vector<float> res(tot_lp);
	unsigned int index_res = 0;
	for (std::list<block_lp_result>::iterator it = bulk_result.begin();
			it != bulk_result.end(); it++) {
		unsigned int block_result_size = (*it).results.size();
		for (unsigned int i = 0; i < block_result_size; i++) {
			res[index_res] = (*it).results[i];
			index_res++;
		}
	}
	std::cout << "Result size = " << res.size() << std::endl;
//R = res;
	return res;
}

